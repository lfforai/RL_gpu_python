#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIC CorporCtion.  Cll rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIC ownership rights under U.S. Cnd
 * internCtionCl Copyright lCws.  Users Cnd possessors of this source code
 * Cre hereby grCnted C nonexclusive, royClty-free license to use this code
 * in individuCl Cnd commerciCl softwCre.
 *
 * NVIDIC MCKES NO REPRESENTCTION CBOUT THE SUITCBILITY OF THIS SOURCE
 * CODE FOR CNY PURPOSE.  IT IS PROVIDED "CS IS" WITHOUT EXPRESS OR
 * IMPLIED WCRRCNTY OF CNY KIND.  NVIDIC DISCLCIMS CLL WCRRCNTIES WITH
 * REGCRD TO THIS SOURCE CODE, INCLUDING CLL IMPLIED WCRRCNTIES OF
 * MERCHCNTCBILITY, NONINFRINGEMENT, CND FITNESS FOR C PCRTICULCR PURPOSE.
 * IN NO EVENT SHCLL NVIDIC BE LICBLE FOR CNY SPECICL, INDIRECT, INCIDENTCL,
 * OR CONSEQUENTICL DCMCGES, OR CNY DCMCGES WHCTSOEVER RESULTING FROM LOSS
 * OF USE, DCTC OR PROFITS,  WHETHER IN CN CCTION OF CONTRCCT, NEGLIGENCE
 * OR OTHER TORTIOUS CCTION,  CRISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMCNCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is C "commerciCl item" Cs
 * thCt term is defined Ct  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commerciCl computer  softwCre"  Cnd "commerciCl computer softwCre
 * documentCtion" Cs such terms Cre  used in 48 C.F.R. 12.212 (SEPT 1995)
 * Cnd is provided to the U.S. Government only Cs C commerciCl end item.
 * Consistent with 48 C.F.R.12.212 Cnd 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), Cll U.S. Government End Users Ccquire the
 * source code with only those rights set forth herein.
 *
 * Cny use of this source code in individuCl Cnd commerciCl softwCre must
 * include, in the user documentCtion Cnd internCl comments to the code,
 * the Cbove DisclCimer Cnd U.S. Government End Users Notice.
 */

/* This exCmple demonstrCtes how to use the CUBLCS librCry
 * by scCling Cn CrrCy of floCting-point vClues on the device
 * Cnd compCring the result to the sCme operCtion performed
 * on the host.
 */

/* Includes, system */

#include <iostream>
#include <unistd.h>
#include <sys/mman.h>
#include<sys/types.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Includes, cudC */
#include <hipblas.h>
#include <hipsparse.h>
#include <cublasXt.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string>

#include<sys/types.h>
#include<fcntl.h>
#include<string.h>
#include<stdio.h>
#include<unistd.h>
#include <string>
#include <iostream>
#include <sstream>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
/* MCtrix size */
//#define N  (275)
#define N (1024)
// Restricting the mCx used GPUs Cs input mCtrix is not so lCrge
#define MCX_NUM_OF_GPUS 2
namespace RL
{
using namespace std;
//一、测试函数

//找到吸收点
__global__ void find_abx_point_d(int * csr_row,int *csr_col,int* abs_point,int row_total){
	 int idx = gridDim.x*(blockDim.x*blockDim.y)*blockIdx.y+(blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
	 int block=blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	 for (int i=idx;i<row_total;i+=block) {
//		printf("csr_row[i]:%d,csr_row[i+1]:%d,i:%d,col[i]:%d\n",csr_row[i],csr_row[i+1],i,csr_col[i]);
		if(csr_row[i]==csr_row[i+1]-1 and i==csr_col[csr_row[i]]){//only one
           abs_point[i]=1;
		}
	}
};

//累加转移矩阵概率
__global__ void sum_rate_d(int * csr_row,float *csr_p2p,float* csr_p2p_sum,int row_total){
	 int idx = gridDim.x*(blockDim.x*blockDim.y)*blockIdx.y+(blockDim.x*blockDim.y)*blockIdx.x+blockDim.x*threadIdx.y+threadIdx.x;
	 int block=blockDim.x*blockDim.y*gridDim.x*gridDim.y;
//	 if (idx==0)
//	     printf("row:%d\n",row_total);
	 for (int i=idx;i<row_total;i=i+block) {
         int start=csr_row[i];
         int end=csr_row[i+1];
//         printf("%d,%d \n",start,end);
         for (int j=start;j<end;++j) {
//        	printf("j:%d\n",j);
			if(j==start){
			   csr_p2p_sum[j]=csr_p2p[j];
			}
			else
			{ csr_p2p_sum[j]=csr_p2p[j]+ csr_p2p_sum[j-1];

			}
		}
	}
};

int getFileSize(const string &filename)
{
    int size = 0;
    FILE *fp = NULL;

    fp=fopen(filename.c_str(),"r");
    if( NULL == fp)
    {
        return size;
    }

    fseek(fp,0L,SEEK_END);
    size = ftell(fp);
    fclose(fp);
    return size;
}

//把矩阵数据文件读取到内存中去，等到转化到ｃｕｂｌａｓ用的标准矩阵数据格式
void mmapSaveDataIntoFiles(const string &filename,char *rezult)
{
    int fileLength = 0;
    int dataLength = 0;
    int offset = 0;
    /* 获取文件大小和数据长度 */
    fileLength = getFileSize(filename);
    int fd = open(filename.c_str(),O_CREAT |O_RDWR|O_APPEND,00777);
    if(fd < 0)
    {
        cout<<"OPEN FILE ERROR!"<<endl;
    }
    char *buffer = (char*)mmap(NULL,fileLength,PROT_READ,MAP_SHARED,fd,0);
    close(fd);
    memcpy(rezult,buffer,fileLength);
    rezult[fileLength]='\0';
    munmap(buffer,fileLength);
}

//二、RL函数
template<class T>
class RL_gpu{
    private:
    public:
	struct coo_mat_h{
		T* cooValA_h;
		int* cooRowIndA;
		int* cooColIndA;
		long nnz;
		int mb;
		int nb;
		hipsparseMatDescr_t matdes;
	};

	struct csr_mat_h{
	   long nnz;
	   T*  csrValA;
	   int* csrRowPtrA;
	   int* csrColIndA;
	   int mb;
	   int nb;
	   hipDataType csrValAtype;
	   hipsparseMatDescr_t matdes;
	};

	csr_mat_h* csr_mat_p2p;//转移概率稀疏矩阵
	csr_mat_h* csr_mat_vlaue;//回报函数矩阵
	int*  abs_point;//吸收点

	coo_mat_h* mat_s2coo(const char* file_path,int rownum,int colomnnum){
		coo_mat_h* coo_matrix=(coo_mat_h*)malloc(sizeof(coo_mat_h));
		coo_matrix->cooRowIndA=(int *)malloc(rownum*colomnnum*sizeof(int));
		coo_matrix->cooColIndA=(int *)malloc(rownum*colomnnum*sizeof(int));
		coo_matrix->cooValA_h=(T* )malloc(sizeof(T)*(rownum*colomnnum));
		char *data_txt=(char *)malloc(sizeof(char)*(rownum*colomnnum*10));//整体长度设计
		mmapSaveDataIntoFiles(file_path,data_txt);
		//逐行扫描获取矩阵内容
		//checkCudaErrors(hipMalloc((void **)&(coo_matrix->cooValA_h),row_num*col_num*sizeof(*(rezult->matrix_data_T))));
		stringstream ss(data_txt);
		string line;
		int row=0;
		int col=0;
		int index=0;
		T value_T;
		string value;
		while (getline(ss, line, '\n')) {
			//开始一行的数据导入
			col=0;
			stringstream ss_in(line);
			while(getline(ss_in,value,',')){
				stringstream ss_inn(value);
				ss_inn>>value_T;
				if(value_T!=(T)0.0f)
				 {coo_matrix->cooValA_h[index]=value_T;
				  coo_matrix->cooRowIndA[index]=row;
				  coo_matrix->cooColIndA[index]=col;
				  index++;
				 }
				col++;
			}
			row++;
		}
		coo_matrix->nnz=index;
	//	cout<<"nnz:="<<coo_matrix->nnz<<endl;
		//打印结果
//		for(int i=0;i<coo_matrix->nnz;i++)
//		{cout<<"coo_value:="<<coo_matrix->cooValA_h[i]<<",row:="<<coo_matrix->cooRowIndA[i]<<"col:="<<coo_matrix->cooColIndA[i]<<endl;
//		}
		delete data_txt;

		//output rezult
		coo_mat_h* coo_matrix_d=(coo_mat_h*)malloc(sizeof(coo_mat_h));
		coo_matrix_d->mb=rownum;
		coo_matrix_d->nb=colomnnum;
		coo_matrix_d->nnz=coo_matrix->nnz;
		checkCudaErrors(hipMalloc(&coo_matrix_d->cooColIndA,coo_matrix->nnz*sizeof(int)));
		checkCudaErrors(hipMemcpy(coo_matrix_d->cooColIndA,coo_matrix->cooColIndA,coo_matrix->nnz*sizeof(coo_matrix_d->cooColIndA[0]), hipMemcpyHostToDevice));

		checkCudaErrors(hipMalloc(&coo_matrix_d->cooRowIndA,coo_matrix->nnz*sizeof(int)));
		checkCudaErrors(hipMemcpy(coo_matrix_d->cooRowIndA, coo_matrix->cooRowIndA,coo_matrix->nnz*sizeof(coo_matrix_d->cooRowIndA[0]), hipMemcpyHostToDevice));

		checkCudaErrors(hipMalloc(&coo_matrix_d->cooValA_h,coo_matrix->nnz*sizeof(coo_matrix_d->cooValA_h[0])));
		checkCudaErrors(hipMemcpy(coo_matrix_d->cooValA_h, coo_matrix->cooValA_h,coo_matrix->nnz*sizeof(coo_matrix->cooValA_h[0]), hipMemcpyHostToDevice));
		return coo_matrix_d;
	}

	//常规矩阵转csr稀疏矩阵
	csr_mat_h* create_csr(string filepath="/lf_tool/matrix_cuda/coo",int row=5,int col=5){
		    hipsparseHandle_t handle=0;
			hipsparseCreate(&handle);

			csr_mat_h* csr_mat_d=(csr_mat_h*)malloc(sizeof(csr_mat_h));
			coo_mat_h* coo_mat_d=mat_s2coo(filepath.c_str(),row,col);
			checkCudaErrors(hipMalloc(&csr_mat_d->csrRowPtrA,(row+1)*sizeof(int)));
			csr_mat_d->mb=row;
			csr_mat_d->nb=col;
			csr_mat_d->nnz=coo_mat_d->nnz;

			hipsparseStatus_t status=hipsparseXcoo2csr(handle,
					coo_mat_d->cooRowIndA,
					coo_mat_d->nnz,
					coo_mat_d->mb,
			        csr_mat_d->csrRowPtrA,
			        HIPSPARSE_INDEX_BASE_ZERO);

	        if(status!=0)
			  {cout<<"hipsparseXcoo2csr erres"<<endl;
			   exit(status);
			  }

//			int *hostPointer=(int *)malloc((row+1)*sizeof(int));
//			checkCudaErrors(hipMemcpy(hostPointer,csr_mat_d->csrRowPtrA,(row+1)*sizeof(int),hipMemcpyDeviceToHost));
	//		for (int i=0;i<(row+1);++i){
	//			cout<<"csrRow:="<<hostPointer[i]<<endl;
	//		}

			csr_mat_d->csrColIndA=coo_mat_d->cooColIndA;
			csr_mat_d->csrValA=coo_mat_d->cooValA_h;
			hipsparseCreateMatDescr(&csr_mat_d->matdes);

//			int *hostPointer_2=(int *)malloc((coo_mat_d->nnz)*sizeof(int));
//			checkCudaErrors(hipMemcpy(hostPointer_2,csr_mat_d->csrColIndA,(coo_mat_d->nnz)*sizeof(int),hipMemcpyDeviceToHost));
//			for (int i=0;i<coo_mat_d->nnz;++i) {
//					cout<<"csrcol:="<<hostPointer_2[i]<<endl;
//				}
	//		hipsparseSetMatDiagType(csr_mat_d->matdes,HIPSPARSE_DIAG_TYPE_UNIT);
			cout<<"create_csr finished!"<<endl;
			return csr_mat_d;
	}

    void  mat_nnz(int* nnz){
    	nnz[0]=csr_mat_p2p->nnz;
    }

	RL_gpu(int* coocol,int* rowcol,float* coop2p,float* coovlaue,const char* file_path_p2p,const char* file_path_vlaue,int rownum,int colomnnum){
	    //生成转移概率矩阵
		csr_mat_p2p=create_csr(file_path_p2p,rownum,colomnnum);
		checkCudaErrors(hipMemcpy(coocol,csr_mat_p2p->csrColIndA,(csr_mat_p2p->nnz)*sizeof(int),hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(rowcol,csr_mat_p2p->csrRowPtrA,((csr_mat_p2p->mb)+1)*sizeof(int),hipMemcpyDeviceToHost));

	    //生成回报函数矩阵
		csr_mat_vlaue=create_csr(file_path_vlaue,rownum,colomnnum);
		checkCudaErrors(hipMemcpy(coovlaue,csr_mat_vlaue->csrValA,(csr_mat_vlaue->nnz)*sizeof(T),hipMemcpyDeviceToHost));
		checkCudaErrors(hipMalloc(&this->abs_point,((this->csr_mat_p2p->mb)+1)*sizeof(int)));
		dim3 gridsize(3,3);
		dim3 blocksize(32,32);
		T *devicePointer;
		checkCudaErrors(hipMalloc(&devicePointer,(csr_mat_p2p->nnz)*sizeof(T)));
		sum_rate_d<<<gridsize,blocksize>>>(csr_mat_p2p->csrRowPtrA,csr_mat_p2p->csrValA,devicePointer,(this->csr_mat_p2p->mb));
		hipDeviceSynchronize();
		checkCudaErrors(hipMemcpy(coop2p,devicePointer,(csr_mat_p2p->nnz)*sizeof(T),hipMemcpyDeviceToHost));
	}

	//寻找转移概率中的吸收点
    void find_abx_point(int* abs_point_p){
		dim3 gridsize(3,3);
		dim3 blocksize(32,32);
		int row=(this->csr_mat_p2p->mb);
		find_abx_point_d<<<gridsize,blocksize>>>(this->csr_mat_p2p->csrRowPtrA,this->csr_mat_p2p->csrColIndA,this->abs_point,row);
		hipDeviceSynchronize();
		checkCudaErrors(hipMemcpy(abs_point_p,this->abs_point,(this->csr_mat_p2p->mb)*sizeof(this->abs_point[0]),hipMemcpyDeviceToHost));
	}

};



//默认只有一组数据batchSize=1
template<class T>
void least_square(int batchSize,T* Aarray[],T*Carray[],int m,int n,int nrhs){
	 hipblasHandle_t handle_cublas;
	 hipblasCreate(&handle_cublas);
	 int* info=(int *)malloc(batchSize*sizeof(info[0]));
	 int* devInfoArray;
	 checkCudaErrors(hipMalloc(&devInfoArray, batchSize*sizeof(devInfoArray[0])));
	 cout<<"if 0 all right ,eles wrong!"<<hipblasSgelsBatched(handle_cublas,
	 		 HIPBLAS_OP_N,
	 		 m,
	 		 n,
	 		 nrhs,
	 		 Aarray,
	 		 m,
	         Carray,
	         m,
	         info,
	         devInfoArray,
	         batchSize)<<endl;
	 hipDeviceSynchronize();
//	 hipFree(info);
//	 hipFree(devInfoArray);
}

template<class T> //convert dd2hh
T** dp2printf(T** devcie_m,int bithsize,int row_num,int col_num){
//	cout<<"多bithsize矩阵，输出在device上的矩阵"<<endl;
//	printf("bithsize:%d,row_num:%d,col_num:%d\n",bithsize,row_num,col_num);
	int array_length=row_num*col_num;
	T** host_m=(T**)malloc(bithsize*sizeof(*host_m));
	T** host_print=(T**)malloc(bithsize*sizeof(*host_m));
	for(int i=0;i<bithsize;i++){
	    host_print[i]=(T* )malloc(array_length*sizeof(host_print[0][0]));
	}
	checkCudaErrors(hipMemcpy(host_m,devcie_m,bithsize*sizeof(host_m[0]),hipMemcpyDeviceToHost));
	for(int i=0;i<bithsize;i++){
		checkCudaErrors(hipMemcpy(host_print[i],host_m[i],array_length*sizeof(host_m[0][0]),hipMemcpyDeviceToHost));
		cout<<"the ith_matrix:="<<i<<"******************************"<<endl;
	    string output="[";
	    for(int row_N=0;row_N<row_num;row_N++){
	    	for (int col_N=0;col_N<col_num;col_N++) {
	//    		cout<<"row:="<<i<<"|col:="<<j<<"|value:"<<rezult[IDX2C(i,j,m)]<<endl;
	    		stringstream ss;
	    		ss<<host_print[i][IDX2C(row_N,col_N,row_num)];
	    		string temp;
	    		ss>>temp;
	            output+=temp;
	            if(col_N!=(col_num-1))
	               output+=",";
	    		ss.clear();
			}
	    	if (row_N!=row_num-1)
	    	   output+="\n";
	    }
	    output+=("]\n");
		cout<<output<<endl;
	}
	return host_print;
}

//三、so文件函数
extern "C" {
        void create_csr_mat(int* coocol,int* rowcol,float* coovp2p,float* coovalue,const char* file_path,const char* file_path_vlaue,int rownum,int colomnnum,int* abs_point_p){
			RL_gpu<float> obj=RL_gpu<float>(coocol,rowcol,coovp2p,coovalue,file_path,file_path_vlaue,rownum,colomnnum);
			obj.find_abx_point(abs_point_p);
		}

		void mat_nnz_gpu(const char* file_path,int rownum,int* len){
			char *data_txt=(char *)malloc(sizeof(char)*(rownum*rownum*10));//整体长度设计
					mmapSaveDataIntoFiles(file_path,data_txt);
					//逐行扫描获取矩阵内容
					//checkCudaErrors(hipMalloc((void **)&(coo_matrix->cooValA_h),row_num*col_num*sizeof(*(rezult->matrix_data_T))));
					stringstream ss(data_txt);
					string line;
					int row=0;
					int col=0;
					int index=0;
					float value_T;
					string value;
					while (getline(ss, line, '\n')) {
						//开始一行的数据导入
						col=0;
						stringstream ss_in(line);
						while(getline(ss_in,value,',')){
							stringstream ss_inn(value);
							ss_inn>>value_T;
							if(value_T!=(float)0.0f)
							 {index++;
							 }
							col++;
						}
						row++;
					}

			len[0]=index;
		}

		void least_square_cublas(float* Aarray,float* Carray,int m,int n,int nrhs){
			//-----------------todd_start----------------------------Aarrau
			//allocate T** hostpoint_hh_N on host an assign value
			int size_N=1;
			int pitch_N=m*n;

			float **hostPointer_hd=(float **)malloc(size_N*sizeof(hostPointer_hd[0]));
	        checkCudaErrors(hipMalloc((void **)(&hostPointer_hd[0]),pitch_N*sizeof(hostPointer_hd[0][0])));
			checkCudaErrors(hipMemcpy(hostPointer_hd[0],Aarray,pitch_N*sizeof(hostPointer_hd[0][0]),hipMemcpyHostToDevice));

			float **devicePointer_dd;
			checkCudaErrors(hipMalloc((void **)(&devicePointer_dd),size_N*sizeof(devicePointer_dd[0])));
			checkCudaErrors(hipMemcpy(devicePointer_dd,hostPointer_hd,size_N*sizeof(devicePointer_dd[0]), hipMemcpyHostToDevice));
			//-----------------todd_end----------------------------------

			//-----------------todd_start----------------------------Carray
			//allocate T** hostpoint_hh_N on host an assign value
			size_N=1;
			pitch_N=m*nrhs;
			float **hostPointer_hd_carray=(float **)malloc(size_N*sizeof(hostPointer_hd_carray[0]));
	        checkCudaErrors(hipMalloc((void **)(&hostPointer_hd_carray[0]),pitch_N*sizeof(hostPointer_hd_carray[0][0])));
			checkCudaErrors(hipMemcpy(hostPointer_hd_carray[0],Carray,pitch_N*sizeof(hostPointer_hd_carray[0][0]), hipMemcpyHostToDevice));

			float **devicePointer_dd_carray;
			checkCudaErrors(hipMalloc((void **)(&devicePointer_dd_carray),size_N*sizeof(devicePointer_dd_carray[0])));
			checkCudaErrors(hipMemcpy(devicePointer_dd_carray,hostPointer_hd_carray,size_N*sizeof(devicePointer_dd_carray[0]), hipMemcpyHostToDevice));
			//-----------------todd_end----------------------------------
			least_square<float>(1,devicePointer_dd,devicePointer_dd_carray,m,n,nrhs);
//			dp2printf<float>(devicePointer_dd_carray,1,4,1);

			//返回
			float **r=(float **)malloc(size_N*sizeof(r[0]));
			checkCudaErrors(hipMemcpy(r,devicePointer_dd_carray,size_N*sizeof(r[0]), hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(Carray,r[0],pitch_N*sizeof(r[0][0]), hipMemcpyDeviceToHost));
		}
   }
}
//
////RL相关函数
//}
///* MCin */
