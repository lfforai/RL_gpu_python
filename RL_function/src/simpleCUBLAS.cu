#include "hip/hip_runtime.h"
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
#include <math.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <typeinfo>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif
#define MIN(a,b) ((a)<(b)? (a):(b))


#include "hipsparse.h"
#include <iostream>
#include <unistd.h>
#include<sys/types.h>
#include<fcntl.h>
#include<string.h>
#include<stdio.h>
#include<unistd.h>
#include <string>
#include <iostream>
#include <unistd.h>
#include <sys/mman.h>
using namespace std;
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define AP_LOWER(i,j,ld) (i-j+((2*ld-j+1)*j)/2)//三角矩阵压缩模式lower
#define AP_UPPER(i,j) (i+(j*(j+1))/2)//三角矩阵压缩模式upper

//----------------------------Frist-file_input------------------------
//一.catch file length
int getFileSize(const string &filename)
{
    int size = 0;
    FILE *fp = NULL;

    fp=fopen(filename.c_str(),"r");
    if( NULL == fp)
    {
        return size;
    }

    fseek(fp,0L,SEEK_END);
    size = ftell(fp);
    fclose(fp);
    return size;
}

//把矩阵数据文件读取到内存中去，等到转化到ｃｕｂｌａｓ用的标准矩阵数据格式
void mmapSaveDataIntoFiles(const string &filename,char *rezult)
{
    int fileLength = 0;
    int dataLength = 0;
    int offset = 0;
    /* 获取文件大小和数据长度 */
    fileLength = getFileSize(filename);
    int fd = open(filename.c_str(),O_CREAT |O_RDWR|O_APPEND,00777);
    if(fd < 0)
    {
        cout<<"OPEN FILE ERROR!"<<endl;
    }
    char *buffer = (char*)mmap(NULL,fileLength,PROT_READ,MAP_SHARED,fd,0);
    close(fd);
    memcpy(rezult,buffer,fileLength);
    rezult[fileLength]='\0';
    munmap(buffer,fileLength);
}

//----------------------------Seconde-cublas------------------------
enum d_or_v{ondevice,onhost};//0返回值在device上，１／返回值在host上

enum matrix_type
{g_packed,//普通矩阵
 g_banded,//gb,普通带状矩阵，需要ｋｌ，ｍｌ表示上下斜线的条数
 symmetric_banded, //sb,对称带状矩阵，ｋ用来表示ｓｕｐｅｒ或ｓｕｂ斜线的条数，hipblasFillMode_t参数储存上下三角
 symmetric_packed,//sp,对称原始矩阵：hipblasFillMode_t
 triangular_packed,//tp,三角原始矩阵：hipblasDiagType_t是否有对角线，hipblasFillMode_t上下三角
 triangular_banded,//tb,三角带状矩阵：hipblasDiagType_t是否有对角线，hipblasFillMode_t上下三角，ｋ用来表示ｓｕｐｅｒ或ｓｕｂ斜线的条数
 Hermitian_packed,//ｈp,自共轭普通矩阵，Ａ＝ＡＨ,hipblasFillMode_t
 Hermitian_banded//hb,自共轭带状矩阵,对称带状矩阵，ｋ用来表示ｓｕｐｅｒ或ｓｕｂ斜线的条数，hipblasFillMode_t参数储存上下三角
};

template<class T>
struct matrix_info{
	T** matrix_data_TT;//*[]
	T*  matrix_data_T;//*
	long  row;
	long  colomn;
    long  idx;
    int   batchCount=1;
    matrix_type type_mat=g_packed;
};

//gpu point be converted to cpu point for printf() using
//return menory in cpu
template<class T> //convert dd2hh
T** dp2printf(T** devcie_m,int bithsize,int row_num,int col_num){
//	cout<<"多bithsize矩阵，输出在device上的矩阵"<<endl;
//	printf("bithsize:%d,row_num:%d,col_num:%d\n",bithsize,row_num,col_num);
	int array_length=row_num*col_num;
	T** host_m=(T**)malloc(bithsize*sizeof(*host_m));
	T** host_print=(T**)malloc(bithsize*sizeof(*host_m));
	for(int i=0;i<bithsize;i++){
	    host_print[i]=(T* )malloc(array_length*sizeof(host_print[0][0]));
	}
	checkCudaErrors(hipMemcpy(host_m,devcie_m,bithsize*sizeof(host_m[0]),hipMemcpyDeviceToHost));
	for(int i=0;i<bithsize;i++){
		checkCudaErrors(hipMemcpy(host_print[i],host_m[i],array_length*sizeof(host_m[0][0]),hipMemcpyDeviceToHost));
		cout<<"the ith_matrix:="<<i<<"******************************"<<endl;
	    string output="[";
	    for(int row_N=0;row_N<row_num;row_N++){
	    	for (int col_N=0;col_N<col_num;col_N++) {
	//    		cout<<"row:="<<i<<"|col:="<<j<<"|value:"<<rezult[IDX2C(i,j,m)]<<endl;
	    		stringstream ss;
	    		ss<<host_print[i][IDX2C(row_N,col_N,row_num)];
	    		string temp;
	    		ss>>temp;
	            output+=temp;
	            if(col_N!=(col_num-1))
	               output+=",";
	    		ss.clear();
			}
	    	if (row_N!=row_num-1)
	    	   output+="\n";
	    }
	    output+=("]\n");
		cout<<output<<endl;
	}
	return host_print;
}

template<class T> //**T 把ｄｅｖｉｃｅ上的内容打印出来
T* dp1_matirx_printf(T* devcie_m,int m,int n,bool ondevice=true){
//	cout<<"输出在device上的矩阵(vector)"<<endl;
	T* rezult;
	if (ondevice==true)
    {rezult=(T*)malloc(n*m*sizeof(rezult[0]));
     checkCudaErrors(hipMemcpy(rezult,devcie_m,m*n*sizeof(rezult[0]),hipMemcpyDeviceToHost));
    }else{
     cout<<"host"<<endl;
     memcpy(rezult,devcie_m,m*n*sizeof(rezult[0]));
    }
    string output="[";
    for(int i=0;i<m;i++){
    	for (int j=0;j<n;j++) {
//    		cout<<"row:="<<i<<"|col:="<<j<<"|value:"<<rezult[IDX2C(i,j,m)]<<endl;
    		stringstream ss;
    		ss<<rezult[IDX2C(i,j,m)];
    		string temp;
    		ss>>temp;
            output+=temp;
            if(j!=(n-1))
               output+=",";
    		ss.clear();
		}
    	if (i!=m-1)
    	   output+="\n";
    }
    output+=("]\n");
    cout<<output<<endl;
	return rezult;
}

//转换为列存储的矩阵g_packed,存储到device上去
template<class T>
matrix_info<T>* m2g_packed(char* data_txt,int batchCount,long row_num,long col_num){
//	cout<<"导入dense矩阵开始"<<endl;
	matrix_info<T>* rezult=(matrix_info<T>*)malloc(sizeof(matrix_info<T>));
	if (batchCount==1)
	   {rezult->row=row_num;
		rezult->colomn=col_num;
		rezult->idx=row_num;
		rezult->batchCount=1;
		T* temp_data=(T*)malloc(row_num*col_num*sizeof(T));//所有分配在ｈｏｓｔ上完成，按需求拷贝到ｄｅｖｉｃｅ
		checkCudaErrors(hipMalloc((void **)&(rezult->matrix_data_T),row_num*col_num*sizeof((rezult->matrix_data_T[0]))));
		long ld=row_num;
		stringstream ss(data_txt);
		string line;
		int row=0;
		int col=0;
		while (getline(ss, line, '\n')) {
			//开始一行的数据导入
			col=0;
			stringstream ss_in(line);
			string value;
			while (getline(ss_in, value, ',')){
				stringstream ss_inn(value);
				ss_inn>>temp_data[IDX2C(row,col,ld)];
				cout<<value<<"|at_matrix[:="<<temp_data[IDX2C(row,col,ld)]<<"]"<<endl;
				col++;
			}
	//		cout<< line << endl;
			++row;
		}
	   int len=row_num*col_num;
//	   if (len>10) len=16;//only out put 10 recored
//	   for(long i=0;i<row_num;i++){
//		   for(long j=0;j<col_num;j++)
//			  {cout<<i<<":="<<temp_data[IDX2C(i,j,ld)]<<endl;}
//		}
	   cout<<"matrix:="<<endl;
	   cout<<data_txt<<endl;
	   checkCudaErrors(hipMemcpy(rezult->matrix_data_T,temp_data,len*sizeof(T),hipMemcpyHostToDevice));
	   }
	   else
	   {    //when  batchCount used
	    	rezult->row=row_num;
			rezult->colomn=col_num;
			rezult->idx=row_num;
			rezult->batchCount=batchCount;

			checkCudaErrors(hipMalloc((void **)&(rezult->matrix_data_TT),batchCount*sizeof(rezult->matrix_data_TT[0])));
			long len=row_num*col_num;
			T** temp_data=(T**)malloc(batchCount*sizeof(*temp_data));//所有分配在ｈｏｓｔ上完成，按需求拷贝到ｄｅｖｉｃｅ
			for(int i=0;i<batchCount;i++){
			   *(temp_data+i)=(T*)malloc(len*sizeof(temp_data[0][0]));
			}

			long ld=row_num;
			stringstream ss(data_txt);
			string line;
			int batch_each=-1;
            int row=0;
			int col=0;
			while (getline(ss, line, '\n')) {
					if(line=="m")
					{//开始一行的数据导入
					 ++batch_each;
					 row=0;
					}else if(line!="#"){
//						cout<<line<<endl;
						col=0;
						stringstream ss_in(line);
						string value;
						while (getline(ss_in, value, ',')){
							stringstream ss_inn(value);
							ss_inn>>temp_data[batch_each][IDX2C(row,col,ld)];
							col++;
						}
						row++;
					}else{
//						printf("g_packed matrix in menory::\n");
//						for(int batch_each=0;batch_each<batchCount;batch_each++)
//						{   cout<<"batch_each:="<<batch_each<<"******************************"<<endl;
//							for(int row=0;row<row_num;row++)
//							{  for(int col=0;col<col_num;col++)
//								  hipsparseMatDescr_t debsrC;
					            //{cout<<"batch_each:"<<batch_each<<"|row:"<<row<<"|col:"<<col<<"|value:="<<temp_data[batch_each][IDX2C(row,col,ld)]<<endl;}
//							}
//						}
						break;
					}
			}

			//copy to device
	        T** temp_2=(T**)malloc(batchCount*sizeof(*temp_2));
	        for(int i=0;i<batchCount;i++){
	            checkCudaErrors(hipMalloc((void **)&(temp_2[i]),len*sizeof(temp_2[0][0])));
	            checkCudaErrors(hipMemcpy(temp_2[i],temp_data[i],len*sizeof(temp_data[0][0]),hipMemcpyHostToDevice));
	        }
			hipMemcpy(rezult->matrix_data_TT,temp_2, batchCount*sizeof(*temp_2),hipMemcpyHostToDevice);
	     }
//	cout<<"导入dense矩阵结束"<<endl;
	return rezult;
}

//symmetric packed matrix,对称方阵
template<class T>
matrix_info<T>* m2g_symmetric_packed(char* data_txt,hipblasFillMode_t uplo=HIPBLAS_FILL_MODE_LOWER,int batchCount=1,long row_num=4){
//	cout<<"导入dense矩阵开始"<<endl;
	matrix_info<T>* rezult=(matrix_info<T>*)malloc(sizeof(matrix_info<T>));
	if (batchCount==1)
	   {rezult->row=row_num;
		rezult->colomn=row_num;
		rezult->idx=row_num;
		rezult->batchCount=1;
		int pitch=(int)(row_num*(row_num+1)/2);
		T* temp_data=(T* )malloc(pitch*sizeof(temp_data[0]));//所有分配在ｈｏｓｔ上完成，按需求拷贝到ｄｅｖｉｃｅ
		checkCudaErrors(hipMalloc((void **)&(rezult->matrix_data_T),pitch*sizeof((rezult->matrix_data_T[0]))));
		long ld=row_num;
		stringstream ss(data_txt);
		string line;
		int row=0;
		int col=0;
		cout<<AP_LOWER(0,1,ld)<<endl;
		while (getline(ss, line, '\n')) {
			//开始一行的数据导入
			col=0;
			stringstream ss_in(line);
			string value;
			while (getline(ss_in, value, ',')){
				if(row-col>=0 and uplo==HIPBLAS_FILL_MODE_LOWER)
				   {stringstream ss_inn(value);
				    ss_inn>>temp_data[AP_LOWER(row,col,ld)];
				    cout<<"row:="<<AP_LOWER(row,col,ld)<<"|value:="<<temp_data[AP_LOWER(row,col,ld)]<<endl;
				   }
				if(row-col<=0 and uplo==HIPBLAS_FILL_MODE_UPPER)
				   {stringstream ss_inn(value);
				    ss_inn>>temp_data[AP_UPPER(row,col)];
				    cout<<"row:="<<AP_UPPER(row,col)<<"|value:="<<temp_data[AP_UPPER(row,col)]<<endl;
				   }
				col++;
			}
	//		cout<< line << endl;
			++row;
		}
		for (int i=0;i<pitch;++i) {
		  cout<<temp_data[i]<<endl;
		}
	   checkCudaErrors(hipMemcpy(rezult->matrix_data_T,temp_data,pitch*sizeof(rezult->matrix_data_T[0]),hipMemcpyHostToDevice));
	   dp1_matirx_printf(rezult->matrix_data_T,1,pitch,true);
	   }
	   else
	   {    //when  batchCount used
	    	rezult->row=row_num;
			rezult->colomn=row_num;
			rezult->idx=row_num;
			rezult->batchCount=batchCount;

			checkCudaErrors(hipMalloc((void **)&(rezult->matrix_data_TT),batchCount*sizeof(rezult->matrix_data_TT[0])));
			long len=(int)(row_num*(row_num+1)/2);
			T** temp_data=(T**)malloc(batchCount*sizeof(*temp_data));//所有分配在ｈｏｓｔ上完成，按需求拷贝到ｄｅｖｉｃｅ
			for(int i=0;i<batchCount;i++){
			   *(temp_data+i)=(T*)malloc(len*sizeof(temp_data[0][0]));
			}

			long ld=row_num;
			stringstream ss(data_txt);
			string line;
			int batch_each=-1;
            int row=0;
			int col=0;
			while (getline(ss, line, '\n')) {
					if(line=="m")
					{//开始一行的数据导入
					 ++batch_each;
					 row=0;
					}else if(line!="#"){
//						cout<<line<<endl;
						col=0;
						stringstream ss_in(line);
						string value;
						while (getline(ss_in, value, ',')){
							stringstream ss_inn(value);
							ss_inn>>temp_data[batch_each][IDX2C(row,col,ld)];
							col++;
						}
						row++;
					}else{
//						printf("g_packed matrix in menory::\n");
//						for(int batch_each=0;batch_each<batchCount;batch_each++)
//						{   cout<<"batch_each:="<<batch_each<<"******************************"<<endl;
//							for(int row=0;row<row_num;row++)
//							{  for(int col=0;col<col_num;col++)
//								  hipsparseMatDescr_t debsrC;
					            //{cout<<"batch_each:"<<batch_each<<"|row:"<<row<<"|col:"<<col<<"|value:="<<temp_data[batch_each][IDX2C(row,col,ld)]<<endl;}
//							}
//						}
						break;
					}
			}

			//copy to device
	        T** temp_2=(T**)malloc(batchCount*sizeof(*temp_2));
	        for(int i=0;i<batchCount;i++){
	            checkCudaErrors(hipMalloc((void **)&(temp_2[i]),len*sizeof(temp_2[0][0])));
	            checkCudaErrors(hipMemcpy(temp_2[i],temp_data[i],len*sizeof(temp_data[0][0]),hipMemcpyHostToDevice));
	        }
			hipMemcpy(rezult->matrix_data_TT,temp_2, batchCount*sizeof(*temp_2),hipMemcpyHostToDevice);
	     }
//	cout<<"导入dense矩阵结束"<<endl;
	return rezult;
}

//这里的带状矩阵都是方阵,row=colomn
//g_banded带状矩阵,symmetric_banded对称带状矩阵(ku设置为０，为sub存储：kｌ设置为０，为super存储)
template<class T>
matrix_info<T>* m2g_banded(char * data_txt,int batchCount=1,long row_num=5,int kl=2,int ku=2){
	matrix_info<T>* rezult=(matrix_info<T>*)malloc(sizeof(matrix_info<T>));
	rezult->row=row_num;
	rezult->colomn=row_num;
	rezult->idx=row_num;
	rezult->batchCount=batchCount;

	if(batchCount==1)
	{T* temp_data=(T*)malloc((kl+ku+1)*row_num*sizeof(T));//所有分配在ｈｏｓｔ上完成，按需求拷贝到ｄｅｖｉｃｅ
	checkCudaErrors(hipMalloc((void **)&(rezult->matrix_data_T),(kl+ku+1)*row_num*sizeof(rezult->matrix_data_T[0])));
	memset(temp_data,0,(kl+ku+1)*row_num*sizeof(T));
	long ld=kl+ku+1;
	stringstream ss(data_txt);
	string line;
	int row=0;
	int col=0;
	T temp_value;
	while (getline(ss, line, '\n')) {
		//开始一行的数据导入
		col=0;
		stringstream ss_in(line);
		string value;
		while (getline(ss_in, value, ',')){
			if(col-row>0 and col-row<=ku)//superline
			  {stringstream ss_inn(value);
			   ss_inn>>temp_value;
			   temp_data[IDX2C(ku-(col-row),col,ld)]=temp_value;
			  }

			if(row-col>0 and row-col<=kl)//sublin
			  {stringstream ss_inn(value);
						ss_inn>>temp_data[IDX2C(row-col+ku,col,ld)];}
			if(row==col)//diagonl
			  {stringstream ss_inn(value);
			   ss_inn>>temp_data[IDX2C(ku,col,ld)];}
//			cout<<value<<"|at_matrix[:="<<IDX2C(row,col,ld)<<"]"<<endl;
			col++;
		}
//		cout<< line << endl;
        ++row;
	}

	//结果输出
	checkCudaErrors(hipMemcpy(rezult->matrix_data_T, temp_data,(kl+ku+1)*row_num*sizeof(T), hipMemcpyHostToDevice));

	//打印结果
	 string output="[";
		for(int i=0;i<ld;i++){//row
			for (int j=0;j<row_num;j++) {//col
	   		//cout<<"row:="<<i<<"|col:="<<j<<"|value:"<<rezult[IDX2C(i,j,m)]<<endl;
				stringstream ss;
				ss<<temp_data[IDX2C(i,j,ld)];
				string temp;
				ss>>temp;
				output+=temp;
				if(j!=(row_num-1))
				   output+=",";
				ss.clear();
			}
			if (i!=ld-1)
			   output+="\n";
		}
		output+=("]\n");
		cout<<output<<endl;
	}else{
		cout<<"cublas is not to suport  batched matrix function!"<<endl;
	}
	return rezult;
}

template<class T>//return2维的在ｄｅｖｉｃｅ上的矩阵
T** cudamenory_create(int batchCount,int array_len){
	T** rezult=(T**)malloc(batchCount*sizeof(*rezult));
	for(int i=0;i<batchCount;i++){
	checkCudaErrors(hipMalloc((void **)&(rezult[i]),array_len*sizeof(rezult[0][0])));
	}
	T** rezult_d;
	checkCudaErrors(hipMalloc((void **)&(rezult_d),batchCount*sizeof(*rezult_d)));
	checkCudaErrors(hipMemcpy(rezult_d,rezult,batchCount*sizeof(*rezult),hipMemcpyHostToDevice));
	return rezult_d;
}


//功能函数封装
//一、cublasgetrfBatched_mat invoke  for matix_LU--start
template<class T>
struct cublasgetrfBatched_mat{
	T **A_value;//output
	T **Carray;
	int *PivotArray;//output
	int *infoArray;//output
	int batchSize;
	int n;

	void init_from_txt(string file_path,long batchsize,long n){
		int char_len=getFileSize(file_path);
		char* matrix_data=(char*)malloc((char_len+20)*sizeof(char));
		mmapSaveDataIntoFiles(file_path,matrix_data);
		matrix_info<T>* A=m2g_packed<T>(matrix_data,batchsize,n,n);
		this->batchSize=A->batchCount;
		this->n=A->row;//A->colomn=A->row;
		this->A_value=A->matrix_data_TT;

		//Carray-----------------todd_start----------------------------
	    //allocate T** hostpoint_hh_N on host an assign value
				int size_N=this->batchSize;
				int pitch_N=(this->n)*(this->n);
				T **hostPointer_hh_N=(T **)malloc(size_N*sizeof(hostPointer_hh_N[0]));
				for (int i= 0;i<size_N;i++) {
					hostPointer_hh_N[i]=(T* )malloc(pitch_N*sizeof(hostPointer_hh_N[0][0]));
						  // assign value to hostPointer[i][j]
					for (int j=0;j<pitch_N;j++) {
						 //assign some value
						hostPointer_hh_N[i][j]=0;
					}
				}
				//allocate T* hd on host  and T** hd on device
				//todd shoud be used frist from the template todd
				//hostPointer_N be created by to hh and  make some value to hh
				T **hostPointer_hd=(T **)malloc(size_N*sizeof(hostPointer_hd[0]));
				for (int i= 0;i<size_N;i++) {
					checkCudaErrors(hipMalloc((void **)(&hostPointer_hd[i]),pitch_N*sizeof(hostPointer_hd[0][0])));
					checkCudaErrors(hipMemcpy(hostPointer_hd[i], hostPointer_hh_N[i],pitch_N*sizeof(hostPointer_hh_N[0][0]), hipMemcpyHostToDevice));
				}
				checkCudaErrors(hipMalloc((void **)(&this->Carray),size_N*sizeof(this->Carray[0])));
				checkCudaErrors(hipMemcpy(this->Carray, hostPointer_hd,size_N*sizeof(this->Carray[0]), hipMemcpyHostToDevice));
		//-----------------todd_end----------------------------------

		//--------*PivotArray
	    checkCudaErrors(hipMalloc(&this->PivotArray, batchsize*n*sizeof(this->PivotArray[0])));
		//-----------------todd_end----------------------------------

		//------------------*infoArray
		checkCudaErrors(hipMalloc(&this->infoArray, batchsize*sizeof(this->infoArray[0])));
        cout<<"inited all right!"<<endl;
	}
};

template<class T>
cublasgetrfBatched_mat<T>* cublasgetrfBatched(string path="/tool-lf/matix_data/QR.txt",long batchsize=3,long n=4,bool print_yorn=true)
{cout<<"cublasgetrfBatched  computing......"<<endl;
 hipblasHandle_t handle_cublas;
 hipblasCreate(&handle_cublas);
 cublasgetrfBatched_mat<T>* getrfBatched_mat=(cublasgetrfBatched_mat<T>* )malloc(sizeof(cublasgetrfBatched_mat<T>));
 getrfBatched_mat->init_from_txt(path,batchsize,n);
 cout<<"compute result is all right? "<<hipblasSgetrfBatched(handle_cublas,
		                           getrfBatched_mat->n,
                                   getrfBatched_mat->A_value,
                                   getrfBatched_mat->n,
                                   getrfBatched_mat->PivotArray,
                                   getrfBatched_mat->infoArray,
                                   getrfBatched_mat->batchSize)<<endl;

  //dp1_matirx_printf(A->matrix_data_T,m,n);
  if (print_yorn==true)
     { cout<<"LU_mat:"<<endl;
	   dp2printf(getrfBatched_mat->A_value,3,4,4);
	   cout<<"PivotArray:"<<endl;
       dp1_matirx_printf(getrfBatched_mat->PivotArray,4,3 );
       cout<<"infoArray:"<<endl;
       dp1_matirx_printf(getrfBatched_mat->infoArray,1,3);
     }
  return getrfBatched_mat;
}

//二、cublasgetrifBatched_mat invoke  for matix_inverse--start
template<class T>
cublasgetrfBatched_mat<T>* cublasgetrifBatched(string path="/tool-lf/matix_data/QR.txt",long batchsize=3,long n=4,bool print_yorn=true)
{cout<<"cublasgetrfBatched  computing......"<<endl;
 hipblasHandle_t handle_cublas;
 hipblasCreate(&handle_cublas);
 cublasgetrfBatched_mat<T>* getrfBatched_mat=(cublasgetrfBatched_mat<T>* )malloc(sizeof(cublasgetrfBatched_mat<T>));
 getrfBatched_mat->init_from_txt(path,batchsize,n);
 cout<<"compute result is all right? "<<hipblasSgetrfBatched(handle_cublas,
		                           getrfBatched_mat->n,
                                   getrfBatched_mat->A_value,
                                   getrfBatched_mat->n,
                                   getrfBatched_mat->PivotArray,
                                   getrfBatched_mat->infoArray,
                                   getrfBatched_mat->batchSize)<<endl;

 cout<<"compute result is all right? "<<hipblasSgetriBatched(handle_cublas,
		                                getrfBatched_mat->n,
		                                getrfBatched_mat->A_value,
		                                getrfBatched_mat->n,
		                                getrfBatched_mat->PivotArray,
		                                getrfBatched_mat->Carray,
		                                getrfBatched_mat->n,
		                                getrfBatched_mat->infoArray,
		                                getrfBatched_mat->batchSize)<<endl;

  //dp1_matirx_printf(A->matrix_data_T,m,n);
  if (print_yorn==true)
     { cout<<"inverse_mat:"<<endl;
	   dp2printf(getrfBatched_mat->Carray,3,4,4);
	   cout<<"PivotArray:"<<endl;
       dp1_matirx_printf(getrfBatched_mat->PivotArray,4,3 );
       cout<<"infoArray:"<<endl;
       dp1_matirx_printf(getrfBatched_mat->infoArray,1,3);
     }
  return getrfBatched_mat;
}

//三、cublasgetrsfBatched_mat invoke  for matix_LU--start，invoking  is false!
template<class T>
struct cublasgetrsfBatched_mat:cublasgetrfBatched_mat<T>{
	T** Barray;
	int nrhs;
	void init_from_txt2(string file_path,long batchsize,long n,long nrhs){
		int char_len=getFileSize(file_path);
		char* matrix_data=(char*)malloc((char_len+20)*sizeof(char));
		mmapSaveDataIntoFiles(file_path,matrix_data);
		matrix_info<T>* A=m2g_packed<T>(matrix_data,batchsize,n,n);
		this->batchSize=A->batchCount;
		this->n=A->row;//A->colomn=A->row;
		this->A_value=A->matrix_data_TT;
		this->nrhs=nrhs;

		//Barray-----------------todd_start----------------------------
		//allocate T** hostpoint_hh_N on host an assign value
		int size_N=this->batchSize;
		int pitch_N=(this->n)*(this->nrhs);
		T **hostPointer_hh_N=(T **)malloc(size_N*sizeof(hostPointer_hh_N[0]));
		for (int i= 0;i<size_N;i++) {
			hostPointer_hh_N[i]=(T* )malloc(pitch_N*sizeof(hostPointer_hh_N[0][0]));
				  // assign value to hostPointer[i][j]
			for (int j=0;j<pitch_N;j++) {
				 //assign some value
				hostPointer_hh_N[i][j]=0;
			}
		}
		//allocate T* hd on host  and T** hd on device
		//todd shoud be used frist from the template todd
		//hostPointer_N be created by to hh and  make some value to hh
		T **hostPointer_hd=(T **)malloc(size_N*sizeof(hostPointer_hd[0]));
		for (int i= 0;i<size_N;i++) {
			checkCudaErrors(hipMalloc((void **)(&hostPointer_hd[i]),pitch_N*sizeof(hostPointer_hd[0][0])));
			checkCudaErrors(hipMemcpy(hostPointer_hd[i], hostPointer_hh_N[i],pitch_N*sizeof(hostPointer_hh_N[0][0]), hipMemcpyHostToDevice));
		}
		checkCudaErrors(hipMalloc((void **)(&this->Barray),size_N*sizeof(this->Barray[0])));
		checkCudaErrors(hipMemcpy(this->Barray, hostPointer_hd,size_N*sizeof(this->Barray[0]), hipMemcpyHostToDevice));
		//-----------------todd_end----------------------------------

		//--------*PivotArray
	    checkCudaErrors(hipMalloc(&this->PivotArray, batchsize*n*sizeof(this->PivotArray[0])));
		//-----------------todd_end----------------------------------

		//------------------*infoArray
	    this->infoArray=(int *)malloc(batchsize*sizeof(this->infoArray[0]));
		cout<<"inited all right!"<<endl;
	}
};

template<class T>
cublasgetrsfBatched_mat<T>* cublasgetrsfBatched(string path="/tool-lf/matix_data/QR.txt",long batchsize=3,long n=4,long nrhs=4,bool print_yorn=true)
{cout<<"cublasgetrsfBatched  computing..cublasmatinvBatched...."<<endl;
 hipblasHandle_t handle_cublas;
 hipblasCreate(&handle_cublas);
 cublasgetrsfBatched_mat<T>* getrsBatched_mat=(cublasgetrsfBatched_mat<T>*)malloc(sizeof(cublasgetrsfBatched_mat<T>));
 getrsBatched_mat->init_from_txt2(path,batchsize,n,nrhs);
 cout<<"compute result is all right? "<<hipblasSgetrsBatched(handle_cublas,
		                           HIPBLAS_OP_N,
		                           getrsBatched_mat->n,
		                           getrsBatched_mat->nrhs,
		                           getrsBatched_mat->A_value,
		                           getrsBatched_mat->n,
		                           getrsBatched_mat->PivotArray,
		                           getrsBatched_mat->Barray,
		                           getrsBatched_mat->n,
		                           getrsBatched_mat->infoArray,
		                           getrsBatched_mat->batchSize)<<endl;

  //dp1_matirx_printf(A->matrix_data_T,m,n);
  if (print_yorn==true)
     { cout<<"LU_mat:"<<endl;
	   dp2printf(getrsBatched_mat->Barray,3,4,4);
	   cout<<"PivotArray:"<<endl;
       dp1_matirx_printf(getrsBatched_mat->PivotArray,4,3);
       cout<<"infoArray:"<<endl;
       dp1_matirx_printf(getrsBatched_mat->infoArray,1,3,false);
     }
  return getrsBatched_mat;
}

//四、less then 32  matrix_inverse
template<class T>
struct cublasmatinvBatched_mat:cublasgetrfBatched_mat<T>{
	T** Ainv;
	void init_from_txt3(string file_path,long batchsize,long n){
		int char_len=getFileSize(file_path);
		char* matrix_data=(char*)malloc((char_len+20)*sizeof(char));
		mmapSaveDataIntoFiles(file_path,matrix_data);
		matrix_info<T>* A=m2g_packed<T>(matrix_data,batchsize,n,n);
		this->batchSize=A->batchCount;
		this->n=A->row;//A->colomn=A->row;
		this->A_value=A->matrix_data_TT;

		//Barray-----------------todd_start----------------------------
		//allocate T** hostpoint_hh_N on host an assign value
		int size_N=this->batchSize;
		int pitch_N=(this->n)*(this->n);
		T **hostPointer_hh_N=(T **)malloc(size_N*sizeof(hostPointer_hh_N[0]));
		for (int i= 0;i<size_N;i++) {
			hostPointer_hh_N[i]=(T* )malloc(pitch_N*sizeof(hostPointer_hh_N[0][0]));
				  // assign value to hostPointer[i][j]
			for (int j=0;j<pitch_N;j++) {
				 //assign some value
				hostPointer_hh_N[i][j]=0;
			}
		}
		//allocate T* hd on host  and T** hd on device
		//todd shoud be used frist from the template todd
		//hostPointer_N be created by to hh and  make some value to hh
		T **hostPointer_hd=(T **)malloc(size_N*sizeof(hostPointer_hd[0]));
		for (int i= 0;i<size_N;i++) {
			checkCudaErrors(hipMalloc((void **)(&hostPointer_hd[i]),pitch_N*sizeof(hostPointer_hd[0][0])));
			checkCudaErrors(hipMemcpy(hostPointer_hd[i], hostPointer_hh_N[i],pitch_N*sizeof(hostPointer_hh_N[0][0]), hipMemcpyHostToDevice));
		}
		checkCudaErrors(hipMalloc((void **)(&this->Ainv),size_N*sizeof(this->Ainv[0])));
		checkCudaErrors(hipMemcpy(this->Ainv, hostPointer_hd,size_N*sizeof(this->Ainv[0]), hipMemcpyHostToDevice));
		//-----------------todd_end----------------------------------

		//------------------*infoArray
		checkCudaErrors(hipMalloc(&this->infoArray, batchsize*sizeof(this->infoArray[0])));
		cout<<"inited all right!"<<endl;
	}
};

template<class T>
cublasmatinvBatched_mat<T>* cublasmatinvBatched(string path="/tool-lf/matix_data/QR.txt",long batchsize=3,long n=4,bool print_yorn=true)
{cout<<"cublasgetrsfBatched  computing......"<<endl;
 hipblasHandle_t handle_cublas;
 hipblasCreate(&handle_cublas);
 cublasmatinvBatched_mat<T>* getrsBatched_mat=(cublasmatinvBatched_mat<T>*)malloc(sizeof(cublasmatinvBatched_mat<T>));
 getrsBatched_mat->init_from_txt3(path,batchsize,n);
 cout<<"compute result is all right? "<<cublasSmatinvBatched(handle_cublas,
			                       getrsBatched_mat->n,
		                           getrsBatched_mat->A_value,
		                           getrsBatched_mat->n,
		                           getrsBatched_mat->Ainv,
		                           getrsBatched_mat->n,
		                           getrsBatched_mat->infoArray,
		                           getrsBatched_mat->batchSize)<<endl;

  //dp1_matirx_printf(A->matrix_data_T,m,n);
  if (print_yorn==true)
     { cout<<"LU_mat:"<<endl;
	   dp2printf(getrsBatched_mat->A_value,3,4,4);
	   cout<<"Ainv:"<<endl;
	   dp2printf(getrsBatched_mat->Ainv,3,4,4);
       cout<<"infoArray:"<<endl;
       dp1_matirx_printf(getrsBatched_mat->infoArray,1,3);
     }
  return getrsBatched_mat;
}

//五、QR分解
template<class T>
struct cublasSgeqrfBatched_mat:cublasgetrfBatched_mat<T>{
	T** TauArray;
	int m;
	void init_from_txt2(string file_path,long batchsize,long n,long m){
		int char_len=getFileSize(file_path);
		char* matrix_data=(char*)malloc((char_len+20)*sizeof(char));
		mmapSaveDataIntoFiles(file_path,matrix_data);
		matrix_info<T>* A=m2g_packed<T>(matrix_data,batchsize,n,m);
		this->batchSize=A->batchCount;
		this->n=A->row;//A->colomn=A->row;
		this->A_value=A->matrix_data_TT;
		this->m=A->colomn;

		//TauArray-----------------todd_start----------------------------
		//allocate T** hostpoint_hh_N on host an assign value
		int size_N=this->batchSize;
		int pitch_N=MIN((this->n),(this->m));
		T **hostPointer_hh_N=(T **)malloc(size_N*sizeof(hostPointer_hh_N[0]));
		for (int i= 0;i<size_N;i++) {
			hostPointer_hh_N[i]=(T* )malloc(pitch_N*sizeof(hostPointer_hh_N[0][0]));
				  // assign value to hostPointer[i][j]
			for (int j=0;j<pitch_N;j++) {
				 //assign some value
				hostPointer_hh_N[i][j]=0;
			}
		}
		//allocate T* hd on host  and T** hd on device
		//todd shoud be used frist from the template todd
		//hostPointer_N be created by to hh and  make some value to hh
		T **hostPointer_hd=(T **)malloc(size_N*sizeof(hostPointer_hd[0]));
		for (int i= 0;i<size_N;i++) {
			checkCudaErrors(hipMalloc((void **)(&hostPointer_hd[i]),pitch_N*sizeof(hostPointer_hd[0][0])));
			checkCudaErrors(hipMemcpy(hostPointer_hd[i], hostPointer_hh_N[i],pitch_N*sizeof(hostPointer_hh_N[0][0]), hipMemcpyHostToDevice));
		}
		checkCudaErrors(hipMalloc((void **)(&this->TauArray),size_N*sizeof(this->TauArray[0])));
		checkCudaErrors(hipMemcpy(this->TauArray, hostPointer_hd,size_N*sizeof(this->TauArray[0]), hipMemcpyHostToDevice));
		//-----------------todd_end----------------------------------

		//------------------*infoArray
		this->infoArray=(int *)malloc(batchsize*sizeof(this->infoArray[0]));
		//checkCudaErrors(hipMalloc(&this->infoArray, batchsize*sizeof(this->infoArray[0])));
		cout<<"inited all right!"<<endl;
	}
};

template<class T>
cublasSgeqrfBatched_mat<T>* hipblasSgeqrfBatched(string path="/tool-lf/matix_data/QR1.txt",long batchsize=3,long n=3,long m=3,bool print_yorn=true)
{cout<<"cublasgetrsfBatched  computing......"<<endl;
 hipblasHandle_t handle_cublas;
 hipblasCreate(&handle_cublas);
 cublasSgeqrfBatched_mat<T>* getrsBatched_mat=(cublasSgeqrfBatched_mat<T>*)malloc(sizeof(cublasSgeqrfBatched_mat<T>));
 getrsBatched_mat->init_from_txt2(path,batchsize,n,m);
 dp2printf(getrsBatched_mat->A_value,batchsize,getrsBatched_mat->n,getrsBatched_mat->m);

 if(typeid(T) == typeid(float))
 {cout<<"compute result is all right? "<<hipblasSgeqrfBatched(handle_cublas,
		                           getrsBatched_mat->n,//row
			                       getrsBatched_mat->m,//col
		                           getrsBatched_mat->A_value,
		                           getrsBatched_mat->n,
		                           getrsBatched_mat->TauArray,
		                  		   getrsBatched_mat->infoArray,
		                           getrsBatched_mat->batchSize)<<endl;}

// if(typeid(T) == typeid(double))
// {cout<<"compute result is all right? "<<hipblasDgeqrfBatched(handle_cublas,
//			                       getrsBatched_mat->n,//row
//			                       getrsBatched_mat->m,//col
//		                           getrsBatched_mat->A_value,
//		                           getrsBatched_mat->n,
//		                           getrsBatched_mat->TauArray,
//		                  		   getrsBatched_mat->infoArray,
//		                           getrsBatched_mat->batchSize)<<endl;}


  if (print_yorn==true)
     { cout<<"R of QR_mat:"<<endl;
	   dp2printf(getrsBatched_mat->A_value,batchsize,getrsBatched_mat->n,getrsBatched_mat->m);
	   cout<<"TauArray:"<<endl;
	   dp2printf(getrsBatched_mat->TauArray,batchsize,1,MIN(getrsBatched_mat->m,getrsBatched_mat->n));
       cout<<"infoArray:"<<endl;
       dp1_matirx_printf(getrsBatched_mat->infoArray,1,batchsize,false);
     }
  return getrsBatched_mat;
}

//六,最小二乘法hipblasSgelsBatched
template<class T>
struct cublasSgelsBatched_mat:cublasgetrfBatched_mat<T>{
	int m;//row>col,m>n
	int nrhs;
    int *info;
	void init_from_txt2(string file_path_x,string file_path_y,long batchsize,long m,long n,long nrhs){
		int char_len=getFileSize(file_path_x);
		char* matrix_data=(char*)malloc((char_len+20)*sizeof(char));

		//input x
		mmapSaveDataIntoFiles(file_path_x,matrix_data);
		matrix_info<T>* A=m2g_packed<T>(matrix_data,batchsize,m,n);//m=row,n=col
		this->batchSize=A->batchCount;
		this->m=A->row;//A->colomn=A->row;
		this->A_value=A->matrix_data_TT;
		this->n=A->colomn;

		//input carray=y
		int char_len_y=getFileSize(file_path_y);
		char* matrix_data_y=(char*)malloc((char_len_y+20)*sizeof(char));
		mmapSaveDataIntoFiles(file_path_y,matrix_data_y);
	    matrix_info<T>* carray=m2g_packed<T>(matrix_data_y,batchsize,m,nrhs);//m=row,n=col,m>n,output n*nrhs
		this->nrhs=nrhs;
		this->Carray=carray->matrix_data_TT;

		//------------------*info
        this->info=(int *)malloc(batchsize*sizeof(this->info[0]));
//	    checkCudaErrors(hipMalloc(&this->info, batchsize*sizeof(this->info[0])));

	    //------------------*infoArray
//		this->infoArray=(int *)malloc(batchsize*sizeof(this->infoArray[0]));
		checkCudaErrors(hipMalloc(&this->infoArray, batchsize*sizeof(this->infoArray[0])));
		cout<<"inited all right!"<<endl;
	}
};

template<class T>
cublasSgelsBatched_mat<T>* hipblasSgelsBatched(string path_x="/tool-lf/matix_data/lq_x.txt",string path_y="/tool-lf/matix_data/lq_y.txt",long batchsize=2,long m=27,long n=4,long nrhs=1,bool print_yorn=true)
{cout<<"cublasgetrsfBatched  computing......"<<endl;
 hipblasHandle_t handle_cublas;
 hipblasCreate(&handle_cublas);
 cublasSgelsBatched_mat<T>* getrsBatched_mat=(cublasSgelsBatched_mat<T>*)malloc(sizeof(cublasSgelsBatched_mat<T>));
 getrsBatched_mat->init_from_txt2(path_x,path_y,batchsize,m,n,nrhs);
// dp2printf(getrsBatched_mat->A_value,batchsize,getrsBatched_mat->m,getrsBatched_mat->n,nrhs);
 if(typeid(T) == typeid(float))
 {cout<<"if 0 all right ,eles wrong!"<<hipblasSgelsBatched(handle_cublas,
		 HIPBLAS_OP_N,
		 getrsBatched_mat->m,
		 getrsBatched_mat->n,
		 getrsBatched_mat->nrhs,
		 getrsBatched_mat->A_value,
		 getrsBatched_mat->m,
         getrsBatched_mat->Carray,
         getrsBatched_mat->m,
         getrsBatched_mat->info,
         getrsBatched_mat->infoArray,
         getrsBatched_mat->batchSize)<<endl;
  }

  if (print_yorn==true)
     { cout<<"A_value of QR_mat:"<<endl;
	   dp2printf(getrsBatched_mat->A_value,batchsize,getrsBatched_mat->m,getrsBatched_mat->n);
	   cout<<"Carray of QR_mat:"<<endl;
	   dp2printf(getrsBatched_mat->Carray,batchsize,getrsBatched_mat->n,getrsBatched_mat->nrhs);
	   cout<<"info:"<<endl;
	   dp1_matirx_printf(getrsBatched_mat->info,1,batchsize,false);
       cout<<"infoArray:"<<endl;
       dp1_matirx_printf(getrsBatched_mat->infoArray,1,batchsize);
     }
  return getrsBatched_mat;
}

//----------------------------Third-cuspares------------------------
template<class T>
struct coo_mat_h{
	T* cooValA_h;
	int* cooRowIndA;
	int* cooColIndA;
	long nnz;
	int mb;
	int nb;
	hipsparseMatDescr_t matdes;
};

template<class T>
struct csr_mat_h{
   long nnz;
   T*  csrValA;
   int* csrRowPtrA;
   int* csrColIndA;
   int mb;
   int nb;
   hipDataType csrValAtype;
   hipsparseMatDescr_t matdes;
};

template<class T>
struct csc_mat_h{
   long nnz;
   T*  cscValA;
   int* cscRowPtrA;
   int* cscColIndA;
   int mb;
   int nb;
   hipsparseMatDescr_t matdes;
};

template<class T>
struct ELL_mat_h{
   hipsparseMatDescr_t matdes;
};

template<class T>
struct HYB_mat_h{
   hipsparseMatDescr_t matdes;
};

template<class T>
struct bsr_mat_h{
   int blockDim=2;
   int mb;
   int nb;
   int nnzb;
   T*  bsrValA;
   int* bsrRowPtrA;
   int* bsrColIndA;
   hipsparseMatDescr_t matdes;
   hipsparseDirection_t dirA;
};

template<class T>
struct bsrx_mat_h{
   int blockDim=2;
   long mb;
   long nb;
   long nnzb;
   T*  bsrValA;
   int* bsrRowPtrA;
   int* bsrEndPtrA;
   int* bsrColIndA;
   hipsparseMatDescr_t matdes;
   hipsparseDirection_t dirA;
};

//host读入txt把一个dense结构的matrix,转化为coo格式的sparse_matrix在cpu上
template<class T>
coo_mat_h<T>* mat_s2coo(const char* file_path,int rownum,int colomnnum){
	coo_mat_h<T>* coo_matrix=(coo_mat_h<T>*)malloc(sizeof(coo_mat_h<T>));
	coo_matrix->cooRowIndA=(int *)malloc(rownum*colomnnum*sizeof(int));
	coo_matrix->cooColIndA=(int *)malloc(rownum*colomnnum*sizeof(int));
	coo_matrix->cooValA_h=(T* )malloc(sizeof(T)*(rownum*colomnnum));
	char *data_txt=(char *)malloc(sizeof(char)*(rownum*colomnnum*10));//整体长度设计
	mmapSaveDataIntoFiles(file_path,data_txt);
	//逐行扫描获取矩阵内容
	//checkCudaErrors(hipMalloc((void **)&(coo_matrix->cooValA_h),row_num*col_num*sizeof(*(rezult->matrix_data_T))));
	stringstream ss(data_txt);
	string line;
	int row=0;
	int col=0;
	int index=0;
	T value_T;
	string value;
	while (getline(ss, line, '\n')) {
		//开始一行的数据导入
		col=0;
		stringstream ss_in(line);
		while(getline(ss_in,value,',')){
			stringstream ss_inn(value);
			ss_inn>>value_T;
			if(value_T!=(T)0.0f)
			 {coo_matrix->cooValA_h[index]=value_T;
			  coo_matrix->cooRowIndA[index]=row;
			  coo_matrix->cooColIndA[index]=col;
			  index++;
			 }
			col++;
		}
		row++;
	}
	coo_matrix->nnz=index;
//	cout<<"nnz:="<<coo_matrix->nnz<<endl;
	//打印结果
//	for(int i=0;i<coo_matrix->nnz;i++)
//	{cout<<"coo_value:="<<coo_matrix->cooValA_h[i]<<",row:="<<coo_matrix->cooRowIndA[i]<<"col:="<<coo_matrix->cooColIndA[i]<<endl;
//	}
	delete data_txt;

	//output rezult
	coo_mat_h<T>* coo_matrix_d=(coo_mat_h<T>*)malloc(sizeof(coo_mat_h<T>));
	coo_matrix_d->mb=rownum;
	coo_matrix_d->nb=colomnnum;
	coo_matrix_d->nnz=coo_matrix->nnz;
	checkCudaErrors(hipMalloc(&coo_matrix_d->cooColIndA,coo_matrix->nnz*sizeof(int)));
	checkCudaErrors(hipMemcpy(coo_matrix_d->cooColIndA,coo_matrix->cooColIndA,coo_matrix->nnz*sizeof(coo_matrix_d->cooColIndA[0]), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&coo_matrix_d->cooRowIndA,coo_matrix->nnz*sizeof(int)));
	checkCudaErrors(hipMemcpy(coo_matrix_d->cooRowIndA, coo_matrix->cooRowIndA,coo_matrix->nnz*sizeof(coo_matrix_d->cooRowIndA[0]), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&coo_matrix_d->cooValA_h,coo_matrix->nnz*sizeof(coo_matrix_d->cooValA_h[0])));
	checkCudaErrors(hipMemcpy(coo_matrix_d->cooValA_h, coo_matrix->cooValA_h,coo_matrix->nnz*sizeof(coo_matrix->cooValA_h[0]), hipMemcpyHostToDevice));
	return coo_matrix_d;
}

//dense vector struct
template<class T>
struct vector_dense_d{
	 int nnz=0;
	 int incx=1;
     T* value_d;
     void init(T* vlaue_h,int incx,long nnz=1){
        checkCudaErrors(hipMalloc((void **)&this->value_d,nnz*sizeof(this->value_d[0])));
		checkCudaErrors(hipMemcpy(this->value_d, vlaue_h,nnz*sizeof(this->value_d[0]), hipMemcpyHostToDevice));
		this->incx=incx;//deflaut=1dp1_matirx_printf
		this->nnz=nnz;//100
     }

     void init_txt(char* txtfile,int incx,long nnz=1){
		T *temp_vector=(T *)malloc(nnz*sizeof(T));
		stringstream ss(txtfile);
		string line;
		int i=0;
		while (getline(ss, line, ',')) {
			stringstream ss_inn(line);
			ss_inn>>temp_vector[i];
			i++;
		}

        checkCudaErrors(hipMalloc((void **)&this->value_d,nnz*sizeof(this->value_d[0])));
		checkCudaErrors(hipMemcpy(this->value_d,temp_vector,nnz*sizeof(this->value_d[0]), hipMemcpyHostToDevice));
		this->incx=incx;//deflaut=1dp1_matirx_printf
		this->nnz=nnz;//100
     }
};

//sparse vector struct
template<class T>
struct vector_sparse_d{
	 long    nnz;//total length of the vector
	 int*  xInd;//index of the vector
     T* value_d;

     void init(T* vlaue_h,int* xInd_h,long nnz){
        checkCudaErrors(hipMalloc((void **)&this->value_d,nnz*sizeof(this->value[0])));
		checkCudaErrors(hipMemcpy(this->value_d, vlaue_h,nnz*sizeof(this->value[0]), hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void**)&this->xInd,nnz*sizeof(xInd_h[0])));
		checkCudaErrors(hipMemcpy(this->xInd, xInd_h,nnz*sizeof(xInd_h[0]), hipMemcpyHostToDevice));
		this->nnz=nnz;
     }

     void init_txt(char* txtfile){
        int j=0;
        stringstream ss(txtfile);
        string line;
        T value;
 		while (getline(ss, line, ',')) {
 				stringstream ss_inn(line);
 				ss_inn>>value;
 			    if (value!=0)
 				   {j++;}
 		}

 		this->nnz=j;//100
        int len=j;
		T* temp_vector=(T*)malloc(j*sizeof(T));
		int* temp_xInd=(int*)malloc(j*sizeof(int));

		ss.str("");
		int i=0;
	    j=0;
	    stringstream ss1(txtfile);
		while (getline(ss1, line, ',')) {
			stringstream ss_inn(line);
			ss_inn>>value;
		    if (value!=0)
			   {temp_vector[i]=value;
			    temp_xInd[i]=j;
//			    cout<<value<<endl;
			    i++;
			   }
		    j++;
		}

        checkCudaErrors(hipMalloc((void **)&this->value_d,this->nnz*sizeof(this->value_d[0])));
		checkCudaErrors(hipMemcpy(this->value_d,temp_vector,this->nnz*sizeof(this->value_d[0]), hipMemcpyHostToDevice));
        checkCudaErrors(hipMalloc((void**)&this->xInd,this->nnz*sizeof(temp_xInd[0])));
		checkCudaErrors(hipMemcpy(this->xInd,temp_xInd,this->nnz*sizeof(temp_xInd[0]), hipMemcpyHostToDevice));

		for (int i=0;i<len;++i) {
				cout<<"i:="<<temp_xInd[i]<<"|value:="<<temp_vector[i]<<endl;
		}
     }
};

template<class T> //coo be converted to csr
csr_mat_h<T>* create_csr(string filepath="/lf_tool/matrix_cuda/coo",int row=5,int col=5){
	    hipsparseHandle_t handle=0;
		hipsparseCreate(&handle);

		csr_mat_h<T>* csr_mat_d=(csr_mat_h<T>*)malloc(sizeof(csr_mat_h<T>));
		coo_mat_h<T>* coo_mat_d=mat_s2coo<T>(filepath.c_str(),row,col);
		checkCudaErrors(hipMalloc(&csr_mat_d->csrRowPtrA,(row+1)*sizeof(int)));
		csr_mat_d->mb=row;
		csr_mat_d->nb=col;
		csr_mat_d->nnz=coo_mat_d->nnz;

		hipsparseStatus_t status=hipsparseXcoo2csr(handle,
				coo_mat_d->cooRowIndA,
				coo_mat_d->nnz,
				5,
		        csr_mat_d->csrRowPtrA,
		        HIPSPARSE_INDEX_BASE_ZERO);

        if(status!=0)
		  {cout<<"hipsparseXcoo2csr erres"<<endl;
		   exit(status);
		  }

		int *hostPointer=(int *)malloc((row+1)*sizeof(int));
		checkCudaErrors(hipMemcpy(hostPointer,csr_mat_d->csrRowPtrA,(row+1)*sizeof(int),hipMemcpyDeviceToHost));
//		for (int i=0;i<(row+1);++i){
//			cout<<"csrRow:="<<hostPointer[i]<<endl;
//		}

		csr_mat_d->csrColIndA=coo_mat_d->cooColIndA;
		csr_mat_d->csrValA=coo_mat_d->cooValA_h;
		hipsparseCreateMatDescr(&csr_mat_d->matdes);

		int *hostPointer_2=(int *)malloc((coo_mat_d->nnz)*sizeof(int));
		checkCudaErrors(hipMemcpy(hostPointer_2,csr_mat_d->csrColIndA,(coo_mat_d->nnz)*sizeof(int),hipMemcpyDeviceToHost));
//		for (int i=0;i<coo_mat_d->nnz;++i) {
//				cout<<"csrcol:="<<hostPointer_2[i]<<endl;
//			}
//		hipsparseSetMatDiagType(csr_mat_d->matdes,HIPSPARSE_DIAG_TYPE_UNIT);
		cout<<"create_csr finished!"<<endl;
		return csr_mat_d;
};

template<class T>
csc_mat_h<T>* csr2csc(csr_mat_h<T>* csr_mat){
hipsparseHandle_t handle=0;
hipsparseCreate(&handle);
csc_mat_h<T>* csc_mat=(csc_mat_h<T>*)malloc(sizeof(csc_mat_h<T>));
csc_mat->mb=csr_mat->mb;
csc_mat->nb=csr_mat->nb;
csc_mat->nnz=csr_mat->nnz;
checkCudaErrors(hipMalloc((void **)&(csc_mat->cscValA),(csr_mat->nnz)*sizeof(csc_mat->cscValA[0])));
checkCudaErrors(hipMalloc((void **)&csc_mat->cscRowPtrA,(csr_mat->nnz)*sizeof(csc_mat->cscRowPtrA[0])));
checkCudaErrors(hipMalloc((void **)&csc_mat->cscColIndA,(csc_mat->nb+1)*sizeof(csc_mat->cscColIndA[0])));
hipsparseStatus_t status=hipsparseScsr2csc(handle,
		                                 csr_mat->mb,
		                                 csr_mat->nb,
		                                 csr_mat->nnz,
		                                 csr_mat->csrValA,
		                                 csr_mat->csrRowPtrA,
		                                 csr_mat->csrColIndA,
		                                 csc_mat->cscValA,
		                                 csc_mat->cscRowPtrA,
		                                 csc_mat->cscColIndA,
		                                 HIPSPARSE_ACTION_SYMBOLIC,
		                                 HIPSPARSE_INDEX_BASE_ZERO);
cout<<hipsparseScsr2csc<<status<<endl;
//dp1_matirx_printf(csc_mat->cscRowPtrA,1,csc_mat->nnz);
//dp1_matirx_printf(csc_mat->cscColIndA,1,csc_mat->nb+1);
}

template<class T>
bsr_mat_h<T>* csr2bsr(csr_mat_h<T>* csr_mat,int blockdim=2){
  hipsparseHandle_t handle=0;
  hipsparseCreate(&handle);
  //block_use
  int m_b=(csr_mat->mb + blockdim-1)/blockdim;
  int n_b=(csr_mat->nb + blockdim-1)/blockdim;

  bsr_mat_h<T>* bsr_mat_rezult=(bsr_mat_h<T>*)malloc(sizeof(bsr_mat_h<T>));
  bsr_mat_rezult->blockDim=blockdim;
  bsr_mat_rezult->mb=csr_mat->mb;
  bsr_mat_rezult->nb=csr_mat->nb;

  hipsparseCreateMatDescr(&bsr_mat_rezult->matdes);
  hipMalloc((void**)&bsr_mat_rezult->bsrRowPtrA, sizeof(int)*(m_b+1));
  bsr_mat_rezult->dirA= HIPSPARSE_DIRECTION_COLUMN;
  hipsparseStatus_t status=hipsparseXcsr2bsrNnz(handle,bsr_mat_rezult->dirA,csr_mat->mb,csr_mat->nb,
		  csr_mat->matdes,csr_mat->csrRowPtrA,csr_mat->csrColIndA,blockdim,
          bsr_mat_rezult->matdes,bsr_mat_rezult->bsrRowPtrA,&(bsr_mat_rezult->nnzb));

  if(status!=0){
	  cout<<"hipsparseXcsr2bsrNnz errose"<<endl;
  }

  hipMalloc((void**)&bsr_mat_rezult->bsrColIndA, sizeof(int)*bsr_mat_rezult->nnzb);
  hipMalloc((void**)&bsr_mat_rezult->bsrValA, sizeof(float)*(blockdim*blockdim)*bsr_mat_rezult->nnzb);

  status=hipsparseScsr2bsr(handle,bsr_mat_rezult->dirA,csr_mat->mb,csr_mat->nb,
		  csr_mat->matdes,csr_mat->csrValA,csr_mat->csrRowPtrA,csr_mat->csrColIndA,blockdim,
          bsr_mat_rezult->matdes
          ,bsr_mat_rezult->bsrValA,bsr_mat_rezult->bsrRowPtrA, bsr_mat_rezult->bsrColIndA);

  if(status!=0){
	  cout<<"hipsparseScsr2bsr errose"<<endl;
  }

  //output bsrrow.
  int b=(bsr_mat_rezult->mb+bsr_mat_rezult->blockDim-1)/bsr_mat_rezult->blockDim+1;
  int *hostPointer=(int *)malloc(b*sizeof(hostPointer[0]));
  checkCudaErrors(hipMemcpy(hostPointer,bsr_mat_rezult->bsrRowPtrA,b*sizeof(hostPointer[0]),hipMemcpyDeviceToHost));

  for (int i=0;i<b;++i) {
	cout<<hostPointer[i]<<endl;
  }
  cout<<"csr2bsr finished!"<<endl;
  return bsr_mat_rezult;
}

//bsr2bsrx
template<class T>
bsrx_mat_h<T>* bsr2bsrx(bsr_mat_h<T>* bsr_mat){
    bsrx_mat_h<T>* rezult=(bsrx_mat_h<T>*)malloc(sizeof(bsrx_mat_h<T>));
    rezult->blockDim=bsr_mat->blockDim;
    rezult->bsrColIndA=bsr_mat->bsrColIndA;
    rezult->bsrValA=bsr_mat->bsrValA;
    rezult->bsrColIndA=bsr_mat->bsrColIndA;
	rezult->dirA=bsr_mat->dirA;
    rezult->matdes=bsr_mat->matdes;
    rezult->mb=bsr_mat->mb;
    rezult->nb=bsr_mat->nb;
    rezult->nnzb=bsr_mat->nnzb;

    //row and rowend
    int b=(rezult->mb+rezult->blockDim-1)/rezult->blockDim+1;
    int *bsrRow_h=(int *)malloc(b*sizeof(bsrRow_h[0]));
    checkCudaErrors(hipMemcpy(bsrRow_h,bsr_mat->bsrRowPtrA,b*sizeof(bsrRow_h[0]),hipMemcpyDeviceToHost));

    checkCudaErrors(hipMalloc(&rezult->bsrRowPtrA,(b-1)*sizeof(rezult->bsrRowPtrA[0])));
    checkCudaErrors(hipMalloc(&rezult->bsrEndPtrA,(b-1)*sizeof(rezult->bsrEndPtrA[0])));

    int* bsrxrow=(int *)malloc((b-1)*sizeof(int));
    int* bsrxend=(int *)malloc((b-1)*sizeof(int));

    for (int i=0;i<b-1;++i) {
    	bsrxrow[i]=bsrRow_h[i];
    	bsrxend[i]=bsrRow_h[i+1];
	}

    checkCudaErrors(hipMemcpy(rezult->bsrRowPtrA,bsrxrow,(b-1)*sizeof(bsrxrow[0]),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rezult->bsrEndPtrA,bsrxend,(b-1)*sizeof(bsrxend[0]),hipMemcpyHostToDevice));
    cout<<"bsrRow:"<<endl;
    dp1_matirx_printf(bsr_mat->bsrRowPtrA,1,b);
    cout<<"bsrxRow:"<<endl;
    dp1_matirx_printf(rezult->bsrRowPtrA,1,b-1);
    cout<<"bsrxEnd:"<<endl;
    dp1_matirx_printf(rezult->bsrEndPtrA,1,b-1);
    return rezult;
}

template<class T> //csr be converted to sr
bsr_mat_h<T>* create_bsr(string filepath="/tool-lf/matix_data/spares_mat",int row=5,int col=5,int blockdim=2){
	csr_mat_h<T>* csr_mat_d=create_csr<T>(filepath,row,col);
	bsr_mat_h<T>* rezult=csr2bsr<T>(csr_mat_d,blockdim);
	return rezult;
}

//create bsrxmv
template<class T>
vector_dense_d<T>* cusparse_bsrmv(string filepath="/lf_tool/matrix_cuda/coo",char * vector="1,1,1,1,1",int m=5,int n=5,int blockdim=2){
    vector_dense_d<T>* rezult=(vector_dense_d<T> *)malloc(sizeof(vector_dense_d<T>));
    hipsparseHandle_t handle=0;
    hipsparseCreate(&handle);

    //create bsr
   	bsr_mat_h<float>* bsr_mat=create_bsr<float>(filepath,m,n,blockdim);
   	hipsparseSetMatDiagType(bsr_mat->matdes,HIPSPARSE_DIAG_TYPE_UNIT);

//   	hipsparseDiagType_t one=hipsparseGetMatDiagType(bsr_mat->matdes);
//   	cout<<one<<endl;
   	//create vector from txt
	vector_dense_d<float>  vector_d;

	int n_b=(bsr_mat->nb+bsr_mat->blockDim-1)/bsr_mat->blockDim;
   	T *temp_vector=(T *)malloc(n_b*bsr_mat->blockDim*sizeof(T));

	stringstream ss(vector);
	string line;
	int i=0;
    while (getline(ss, line, ',')) {
		stringstream ss_inn(line);
	    ss_inn>>temp_vector[i];
	    i++;
    }

    if(i<n_b*bsr_mat->blockDim){
    	for (int index=0;index<n_b*bsr_mat->blockDim-i;++index) {
    		temp_vector[i+index]=0;
		}
    }

   	T beta=0.0;
   	T alpha=1.0;

   	vector_d.init(temp_vector,1,n_b*bsr_mat->blockDim);
   	hipsparseStatus_t status=hipsparseSbsrmv(handle,bsr_mat->dirA,HIPSPARSE_OPERATION_NON_TRANSPOSE,bsr_mat->mb,bsr_mat->nb, bsr_mat->nnzb, &alpha,
       bsr_mat->matdes, bsr_mat->bsrValA, bsr_mat->bsrRowPtrA,bsr_mat->bsrColIndA,bsr_mat->blockDim,vector_d.value_d, &beta,vector_d.value_d);
    cout<<"hipsparseSbsrmv status:="<<status<<endl;
    dp1_matirx_printf(vector_d.value_d,1,n_b*bsr_mat->blockDim);
    return rezult;
}


//create bsrxmv
template<class T>
vector_dense_d<T>* cusparse_bsrxmv(string filepath,char* vector,int m,int n,int blockdim,int sizeOfMask_h,int* bsrMaskPtr_h,int* bsrRowPtrA_hh,int* bsrEndPtrA_hh)
{
    vector_dense_d<T>* rezult=(vector_dense_d<T> *)malloc(sizeof(vector_dense_d<T>));
    hipsparseHandle_t handle=0;
    hipsparseCreate(&handle);

    //create bsr
   	bsr_mat_h<T>* bsr_mat=create_bsr<T>(filepath,m,n,blockdim);
   	bsrx_mat_h<T>* bsrx_mat=bsr2bsrx<T>(bsr_mat);

   	//create vector from txt
	vector_dense_d<T>  vector_d;

	int n_b=(bsrx_mat->nb+bsrx_mat->blockDim-1)/bsrx_mat->blockDim;
   	T *temp_vector=(T *)malloc(n_b*bsrx_mat->blockDim*sizeof(T));

	stringstream ss(vector);
	string line;
	int i=0;
    while (getline(ss, line, ',')) {
		stringstream ss_inn(line);
	    ss_inn>>temp_vector[i];
	    i++;
    }

    if(i<n_b*bsrx_mat->blockDim){
    	for (int index=0;index<n_b*bsr_mat->blockDim-i;++index) {
    		temp_vector[i+index]=0;
		}
    }

   	T beta=0.0;
   	T alpha=1.0;
   	int sizeOfMask=sizeOfMask_h;
   	int* bsrMaskPtr=bsrMaskPtr_h;

   	int *bsrMaskPtr_d;
	checkCudaErrors(hipMalloc(&bsrMaskPtr_d, sizeOfMask*sizeof(bsrMaskPtr[0])));
	checkCudaErrors(hipMemcpy(bsrMaskPtr_d,bsrMaskPtr,sizeOfMask*sizeof(bsrMaskPtr[0]), hipMemcpyHostToDevice));

	//redefine matrix shape
	int* bsrRowPtrA_h=bsrRowPtrA_hh;
	int* bsrEndPtrA_h=bsrEndPtrA_hh;
	checkCudaErrors(hipMemcpy(bsrx_mat->bsrRowPtrA,bsrRowPtrA_h,n_b*sizeof(bsrRowPtrA_h[0]),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(bsrx_mat->bsrEndPtrA,bsrEndPtrA_h,n_b*sizeof(bsrEndPtrA_h[0]),hipMemcpyHostToDevice));

   	vector_d.init(temp_vector,1,n_b*bsrx_mat->blockDim);
   	hipsparseStatus_t status=hipsparseSbsrxmv(handle,bsrx_mat->dirA,HIPSPARSE_OPERATION_NON_TRANSPOSE,sizeOfMask,bsrx_mat->mb,bsrx_mat->nb, bsrx_mat->nnzb, &alpha,
       bsrx_mat->matdes, bsrx_mat->bsrValA,bsrMaskPtr_d,bsrx_mat->bsrRowPtrA,bsrx_mat->bsrEndPtrA,bsrx_mat->bsrColIndA,bsrx_mat->blockDim,vector_d.value_d, &beta,vector_d.value_d);

   	cout<<" hipsparseSbsrxmv status:="<<status<<endl;
    dp1_matirx_printf(vector_d.value_d,1,n_b*bsrx_mat->blockDim);
    return rezult;
}


template<class T>
void test(string filepath="/lf_tool/matrix_cuda/coo",int row=5,int col=5){
	// Suppose that L is m x m sparse matrix represented by CSR format,
	// L is lower triangular with unit diagonal.
	// Assumption:
	// - dimension of matrix L is m,
	// - matrix L has nnz number zero elements,
	// - handle is already created by hipsparseCreate(),
	// - (d_csrRowPtr, d_csrColInd, d_csrVal) is CSR of L on device memory,
	// - d_x is right hand side vector on device memory,
	// - d_y is solution vector on device memory.
    hipsparseHandle_t handle=0;
    hipsparseCreate(&handle);
	csr_mat_h<T>* csr_mat_d=create_csr<T>(filepath,row,col);

	hipsparseMatDescr_t descr =csr_mat_d->matdes;
	csrsv2Info_t info = 0;
	int pBufferSize;
	void *pBuffer = 0;
	int structural_zero;
	int numerical_zero;
	const double alpha = 1.;
	const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	const hipsparseOperation_t trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;

	// step 1: create a descriptor which contains
	// - matrix L is base-1
	// - matrix L is lower triangular
	// - matrix L has unit diagonal, specified by parameter HIPSPARSE_DIAG_TYPE_UNIT
	//   (L may not have all diagonal elements.)
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipDeviceSynchronize();

	// step 2: create a empty info structure
	hipsparseCreateCsrsv2Info(&info);


	// step 3: query how much memory used in csrsv2, and allocate the buffer
	hipsparseDcsrsv2_bufferSize(handle, trans,csr_mat_d->mb,csr_mat_d->nnz, descr,
			csr_mat_d->csrValA,csr_mat_d->csrRowPtrA,csr_mat_d->csrColIndA,info,&pBufferSize);
	// pBuffer returned by hipMalloc is automatically aligned to 128 bytes.

	hipMalloc((void**)&pBuffer, pBufferSize);

    //cout<<"hipsparseDcsrsv2_bufferSize finished"<<endl;
	// step 4: perform analysis
	hipsparseDcsrsv2_analysis(handle, trans,csr_mat_d->mb,csr_mat_d->nnz, descr,
			csr_mat_d->csrValA,csr_mat_d->csrRowPtrA,csr_mat_d->csrColIndA,
	        info, policy, pBuffer);

	// L has unit diagonal, so no structural zero is reported.
	hipsparseStatus_t status = hipsparseXcsrsv2_zeroPivot(handle, info, &structural_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
	   printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
	}

	// step 5: solve L*y = x
	vector_dense_d<double> d_x;
	d_x.init_txt("1,1,1,1,1",1,5);
	vector_dense_d<double> d_y;
	d_y.init_txt("1,1,1,1,1",1,5);

	hipsparseDcsrsv2_solve(handle,trans,csr_mat_d->mb,csr_mat_d->nnz,&alpha,descr,
			csr_mat_d->csrValA,csr_mat_d->csrRowPtrA,csr_mat_d->csrColIndA, info,
	   d_x.value_d, d_y.value_d, policy, pBuffer);

    dp1_matirx_printf<double>(d_y.value_d,1,5);
//	cout<<"hipsparseDcsrsv2_solve finished"<<endl;
	// L has unit diagonal, so no numerical zero is reported.
	status = hipsparseXcsrsv2_zeroPivot(handle, info, &numerical_zero);
	if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
	   printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
	}

//	cout<<"hipsparseDcsrsv2_bufferSize finished"<<endl;
	// step 6: free resources
	hipFree(pBuffer);
	hipsparseDestroyCsrsv2Info(info);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(handle);
	cout<<"test finished"<<endl;
}

//template<class T>
//void cusparseXgemvi(string filepath="/lf_tool/matrix_cuda/coo",int row=5,int col=5){
//	int len=getFileSize(filepath);
//	char* data=(char*)malloc((len+10)*sizeof(char));
//	mmapSaveDataIntoFiles(filepath,data);
//	matrix_info<T>* matrix_packed=m2g_packed<float>(data,1,5,5);
////  matrix_packed->matrix_data_T;
//	vector_sparse_d<float> x_
//
//	The general procedure is as follows:
//
//	int baseC, nnzC;d;
//	x_d.init_txt("1,0,1,0,1");
//	vector_dense_d<float> y_d;
//	y_d.init_txt("0,0,0,0,0",1,5);
//
//    hipsparseHandle_t handle=0;
//    hipsparseCreate(&handle);
//    T alpha=1.0;
//    T beta=0.0;
//    int pBufferSize;
//
//    hipsparseStatus_t status=hipsparseSgemvi_bufferSize(handle,
//    		                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
//    		                  matrix_packed->row,
//    		                  matrix_packed->row,
//    		                  x_d.nnz,
//                              &pBufferSize);
//
//    cout<<"pBufferSize:="<<pBufferSize<<endl;
//
//    void* pBuffer=0;
//    hipMalloc((void**)&pBuffer, pBufferSize);
//
//
//    The general procedure is as follows:
//
//    int baseC, nnzC;
//    cout<<"status"<<status<<endl;
//    status=hipsparseSgemvi(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
//    		       matrix_packed->row,matrix_packed->row,&alpha,
//    		       matrix_packed->matrix_data_T,
//    		       matrix_packed->row,x_d.nnz,
//                   x_d.value_d,
//                   x_d.xInd,
//                   &beta,
//                   y_d.value_d,
//                   HIPSPARSE_INDEX_BASE_ZERO,
//                   pBuffer);
//
//    cout<<"status"<<status<<endl;
//    dp1_matirx_printf(y_d.value_d,1,5);
//}


//a=r*r^T,wan
template<class T>
void csric0(string filepath="/lf_tool/matrix_cuda/coo",int row=5,int col=5){

	csr_mat_h<T>* csr_mat=create_csr<T>(filepath,row,col);

    hipsparseHandle_t handle=0;
    hipsparseCreate(&handle);

    dp1_matirx_printf(csr_mat->csrColIndA,1,csr_mat->nnz);
    hipsparseSetMatType(csr_mat->matdes,HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
    hipsparseSetMatFillMode(csr_mat->matdes,HIPSPARSE_FILL_MODE_LOWER);

    cusparseSolveAnalysisInfo_t info;
    cusparseCreateSolveAnalysisInfo(&info);
	hipsparseStatus_t status=cusparseScsrsv_analysis(handle,
			                HIPSPARSE_OPERATION_NON_TRANSPOSE,
	                        csr_mat->mb,
	                        csr_mat->nnz,
	                        csr_mat->matdes,
	                        csr_mat->csrValA,
	                        csr_mat->csrRowPtrA,
	                        csr_mat->csrColIndA,
	                        info);
//	hipDeviceSynchronize();
    cout<<"cusparseScsrsv_analysis:"<<status<<endl;

	status=cusparseScsric0(handle,
			        HIPSPARSE_OPERATION_NON_TRANSPOSE,
			        csr_mat->mb,
			        csr_mat->matdes,
                    csr_mat->csrValA,
                    csr_mat->csrRowPtrA,
                    csr_mat->csrColIndA,
                    info);
//	hipDeviceSynchronize();
    cout<<"cusparseScsric0:"<<status<<endl;
    dp1_matirx_printf(csr_mat->csrValA,1,csr_mat->nnz);
    dp1_matirx_printf(csr_mat->csrColIndA,1,csr_mat->nnz);
}

//a=r*r^T,wan
template<class T>
void csric02(string filepath="/lf_tool/matrix_cuda/coo_2",int row=5,int col=5){

	csr_mat_h<T>* csr_mat=create_csr<T>(filepath,row,col);
    hipsparseHandle_t handle=0;
    hipsparseCreate(&handle);

    int structural_zero;
    int numerical_zero;
    const hipsparseSolvePolicy_t policy_M  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;

    hipsparseSetMatType(csr_mat->matdes,HIPSPARSE_MATRIX_TYPE_GENERAL);
//    hipsparseSetMatFillMode(csr_mat->matdes,HIPSPARSE_FILL_MODE_LOWER);
    dp1_matirx_printf(csr_mat->csrValA,1,csr_mat->nnz);
    dp1_matirx_printf(csr_mat->csrColIndA,1,csr_mat->nnz);
    dp1_matirx_printf(csr_mat->csrRowPtrA,1,csr_mat->mb+1);

    csric02Info_t info;
    hipsparseCreateCsric02Info(&info);

    int pBufferSize=0;
    void *pBuffer = 0;
    hipsparseStatus_t status=hipsparseScsric02_bufferSize(handle,
                        csr_mat->mb,
                        csr_mat->nnz,
                        csr_mat->matdes,
                        csr_mat->csrValA,
                        csr_mat->csrRowPtrA,
                        csr_mat->csrColIndA,
                        info,
                        &pBufferSize);

    checkCudaErrors(hipMalloc((void**)&pBuffer, pBufferSize));
    hipsparseScsric02_analysis(handle,csr_mat->mb, csr_mat->nnz, csr_mat->matdes,
    		csr_mat->csrValA,csr_mat->csrRowPtrA,csr_mat->csrColIndA,info,
        policy_M, pBuffer);

    status = hipsparseXcsric02_zeroPivot(handle, info, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
       printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    // step 5: M = L * L'
    hipsparseScsric02(handle,csr_mat->mb, csr_mat->nnz, csr_mat->matdes,
    		csr_mat->csrValA,csr_mat->csrRowPtrA,csr_mat->csrColIndA,info,
        policy_M, pBuffer);

    status = hipsparseXcsric02_zeroPivot(handle, info, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
       printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    //hipDeviceSynchronize();
    cout<<"cusparseScsric0:"<<status<<endl;
    dp1_matirx_printf(csr_mat->csrValA,1,csr_mat->nnz);
    dp1_matirx_printf(csr_mat->csrColIndA,1,csr_mat->nnz);
    dp1_matirx_printf(csr_mat->csrRowPtrA,1,csr_mat->mb+1);

}




/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include csrmvmp_example.cpp
 *   g++ -fopenmp -o csrmvmp_example csrmvmp_example.o -L/usr/local/cuda/lib64 -lcublas -lcusparse -lcudart
 *
 */

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

void eigenvalue()
{
      hipblasHandle_t cublasH = NULL;
      hipsparseHandle_t cusparseH = NULL;
      hipStream_t stream = NULL;
      hipsparseMatDescr_t descrA = NULL;

      hipblasStatus_t cublasStat = HIPBLAS_STATUS_SUCCESS;
      hipsparseStatus_t cusparseStat = HIPSPARSE_STATUS_SUCCESS;
      hipError_t cudaStat1 = hipSuccess;
      hipError_t cudaStat2 = hipSuccess;
      hipError_t cudaStat3 = hipSuccess;
      hipError_t cudaStat4 = hipSuccess;
      hipError_t cudaStat5 = hipSuccess;
      const int n = 4;
      const int nnzA = 9;
/*
 *      |    1     0     2     3   |
 *      |    0     4     0     0   |
 *  A = |    5     0     6     7   |
 *      |    0     8     0     9   |
 *
 * eigevales are { -0.5311, 7.5311, 9.0000, 4.0000 }
 *
 * The largest eigenvaluse is 9 and corresponding eigenvector is
 *
 *      | 0.3029  |
 * v =  |     0   |
 *      | 0.9350  |
 *      | 0.1844  |
 */
        const int csrRowPtrA[n+1] = { 0, 3, 4, 7, 9 };
        const int csrColIndA[nnzA] = {0, 2, 3, 1, 0, 2, 3, 1, 3 };
        const double csrValA[nnzA] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0 };
        const double lambda_exact[n] = { 9.0000, 7.5311, 4.0000, -0.5311 };
        const double x0[n] = {1.0, 2.0, 3.0, 4.0 }; /* initial guess */
        double x[n]; /* numerical eigenvector */

        int *d_csrRowPtrA = NULL;
        int *d_csrColIndA = NULL;
        double *d_csrValA = NULL;

        double *d_x = NULL; /* eigenvector */
        double *d_y = NULL; /* workspace */

        const double tol = 1.e-6;
        const int max_ites = 30;

        const double h_one  = 1.0;
        const double h_zero = 0.0;

        printf("example of csrmv_mp \n");
        printf("tol = %E \n", tol);
        printf("max. iterations = %d \n", max_ites);

        printf("1st eigenvaluse is %f\n", lambda_exact[0] );
        printf("2nd eigenvaluse is %f\n", lambda_exact[1] );

        double alpha = lambda_exact[1]/lambda_exact[0] ;
        printf("convergence rate is %f\n", alpha );

        double est_iterations = log(tol)/log(alpha);
        printf("# of iterations required is %d\n", (int)ceil(est_iterations)  );

    /* step 1: create cublas/cusparse handle, bind a stream */
        cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        assert(hipSuccess == cudaStat1);

        cublasStat = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublasStat);

        cublasStat = hipblasSetStream(cublasH, stream);
        assert(HIPBLAS_STATUS_SUCCESS == cublasStat);

        cusparseStat = hipsparseCreate(&cusparseH);
        assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

        cusparseStat = hipsparseSetStream(cusparseH, stream);
        assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

    /* step 2: configuration of matrix A */
        cusparseStat = hipsparseCreateMatDescr(&descrA);
        assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL );

        /* step 3: copy A and x0 to device */
            cudaStat1 = hipMalloc ((void**)&d_csrRowPtrA, sizeof(int) * (n+1) );
            cudaStat2 = hipMalloc ((void**)&d_csrColIndA, sizeof(int) * nnzA );
            cudaStat3 = hipMalloc ((void**)&d_csrValA   , sizeof(double) * nnzA );
            cudaStat4 = hipMalloc ((void**)&d_x         , sizeof(double) * n );
            cudaStat5 = hipMalloc ((void**)&d_y         , sizeof(double) * n );
            assert(hipSuccess == cudaStat1);
            assert(hipSuccess == cudaStat2);
            assert(hipSuccess == cudaStat3);
            assert(hipSuccess == cudaStat4);
            assert(hipSuccess == cudaStat5);

            cudaStat1 = hipMemcpy(d_csrRowPtrA, csrRowPtrA, sizeof(int) * (n+1)   , hipMemcpyHostToDevice);
            cudaStat2 = hipMemcpy(d_csrColIndA, csrColIndA, sizeof(int) * nnzA    , hipMemcpyHostToDevice);
            cudaStat3 = hipMemcpy(d_csrValA   , csrValA   , sizeof(double) * nnzA , hipMemcpyHostToDevice);
            assert(hipSuccess == cudaStat1);
            assert(hipSuccess == cudaStat2);
            assert(hipSuccess == cudaStat3);


        /*
         * step 4: power method
         */
            double lambda = 0.0;
            double lambda_next = 0.0;

        /*
         *  4.1: initial guess x0
         */
            cudaStat1 = hipMemcpy(d_x, x0, sizeof(double) * n, hipMemcpyHostToDevice);
            assert(hipSuccess == cudaStat1);

            for(int ite = 0 ; ite < max_ites ; ite++ ){
        /*
         *  4.2: normalize vector x
         *      x = x / |x|
         */
                double nrm2_x;
                cublasStat = hipblasDnrm2(cublasH,
                                            n,
                                            d_x,
                                            1, // incx,
                                            &nrm2_x  /* host pointer */
                                           );

                assert(HIPBLAS_STATUS_SUCCESS == cublasStat);

                double one_over_nrm2_x = 1.0 / nrm2_x;
                cublasStat = hipblasDscal( cublasH,
                                             n,
                                             &one_over_nrm2_x,  /* host pointer */
                                             d_x,
                                             1 // incx
                                            );
                assert(HIPBLAS_STATUS_SUCCESS == cublasStat);
                /*
                 *  4.3: y = A*x
                 */
                        cusparseStat = cusparseDcsrmv_mp(cusparseH,
                                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                         n,
                                                         n,
                                                         nnzA,
                                                         &h_one,
                                                         descrA,
                                                         d_csrValA,
                                                         d_csrRowPtrA,
                                                         d_csrColIndA,
                                                         d_x,
                                                         &h_zero,
                                                         d_y);
                        assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

                /*
                 *  4.4: lambda = y**T*x
                 */
                        cublasStat = hipblasDdot ( cublasH,
                                                     n,
                                                     d_x,
                                                     1, // incx,
                                                     d_y,
                                                     1, // incy,
                                                     &lambda_next  /* host pointer */
                                                   );
                        dp1_matirx_printf(d_x,1,4);
                        dp1_matirx_printf(d_y,1,4);
                        cout<<lambda_next<<endl;
                        assert(HIPBLAS_STATUS_SUCCESS == cublasStat);

                        double lambda_err = fabs( lambda_next - lambda_exact[0] );
                        printf("ite %d: lambda = %f, error = %E\n", ite, lambda_next, lambda_err );
                /*
                 *  4.5: check if converges
                 */
                        if ( (ite > 0) &&
                             fabs( lambda - lambda_next ) < tol
                        ){
                            break; // converges
                        }

                /*
                 *  4.6: x := y
                 *       lambda = lambda_next
                 *
                 *  so new approximation is (lambda, x), x is not normalized.
                 */
                        cudaStat1 = hipMemcpy(d_x, d_y, sizeof(double) * n , hipMemcpyDeviceToDevice);
                        assert(hipSuccess == cudaStat1);

                        lambda = lambda_next;
                    }
            /*
             * step 5: report eigen-pair
             */
                cudaStat1 = hipMemcpy(x, d_x, sizeof(double) * n, hipMemcpyDeviceToHost);
                assert(hipSuccess == cudaStat1);

                printf("largest eigenvalue is %E\n", lambda );

                printf("eigenvector = (matlab base-1)\n");
                printMatrix(n, 1, x, n, "V0");
                printf("=====\n");


            /* free resources */
                if (d_csrRowPtrA  ) hipFree(d_csrRowPtrA);
                if (d_csrColIndA  ) hipFree(d_csrColIndA);
                if (d_csrValA     ) hipFree(d_csrValA);
                if (d_x           ) hipFree(d_x);
                if (d_y           ) hipFree(d_y);

                if (cublasH       ) hipblasDestroy(cublasH);
                if (cusparseH     ) hipsparseDestroy(cusparseH);
                if (stream        ) hipStreamDestroy(stream);
                if (descrA        ) hipsparseDestroyMatDescr(descrA);

                hipDeviceReset();
}


template<class T>
void eigenvalue2(string file_path="/lf_tool/matrix_cuda/eigenvalue",int row=5)
{
      hipblasHandle_t cublasH = NULL;
      hipsparseHandle_t cusparseH = NULL;
//      hipStream_t stream = NULL;
//      hipsparseMatDescr_t descrA = NULL;

/*
 *      |    1     0     2     3   |
 *      |    0     4     0     0   |
 *  A = |    5     0     6     7   |
 *      |    0     8     0     9   |
 *
 * eigevales are { -0.5311, 7.5311, 9.0000, 4.0000 }
 *
 * The largest eigenvaluse is 9 and corresponding eigenvector is
 *
 *      | 0.3029  |
 * v =  |     0   |
 *      | 0.9350  |
 *      | 0.1844  |
 */
      csr_mat_h<T>* csr_mat=create_csr<T>(file_path,row,row);

      T* v=(T*)malloc(row*sizeof(T));
      for(int i=0;i<row;++i) {
		v[i]=i;
	  }

      T* v_d;
	  checkCudaErrors(hipMalloc((void**)&v_d,row*sizeof(v_d[0])));
	  checkCudaErrors(hipMemcpy(v_d, v,row*sizeof(T), hipMemcpyHostToDevice));

	  size_t bufferSizeInBytes=0;
	  T alpha=1.0;
      T beta=0.0;

      hipsparseStatus_t status;
	  status=cusparseCsrmvEx_bufferSize(cusparseH,
			                           CUSPARSE_ALG_MERGE_PATH,
			                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
			                           csr_mat->mb,
			                           csr_mat->nb,
			                           csr_mat->nnz,
	                                   &alpha,HIP_R_32F,
									   csr_mat->matdes,
									   csr_mat->csrValA,HIP_R_32F,
									   csr_mat->csrRowPtrA,
									   csr_mat->csrColIndA,
									   v_d,HIP_R_32F,
									   &beta,HIP_R_32F,
									   v_d,HIP_R_32F,
									   HIP_R_32F,
									   &bufferSizeInBytes);

     cout<<"cusparseCsrmvEx_bufferSize status:="<<status<<endl;

     void *buffer;
     checkCudaErrors(hipMalloc((void **)&buffer,bufferSizeInBytes));

	 status=cusparseCsrmvEx(cusparseH,
			               CUSPARSE_ALG_MERGE_PATH,
						   HIPSPARSE_OPERATION_NON_TRANSPOSE,
						   csr_mat->mb,
						   csr_mat->nb,
						   csr_mat->nnz,
						   &alpha,HIP_R_32F,
						   csr_mat->matdes,
						   csr_mat->csrValA,HIP_R_32F,
						   csr_mat->csrRowPtrA,
						   csr_mat->csrColIndA,
						   v_d,HIP_R_32F,
						   &beta,HIP_R_32F,
						   v_d,HIP_R_32F,
						   HIP_R_32F,
                           buffer);

	  cout<<"cusparseCsrmvEx status:="<<status<<endl;
      dp1_matirx_printf(v_d,1,row);
      hipDeviceReset();
}

////////////////////////////////////////////////bsrRowPtrC////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    cout<<"---------------------"<<endl;
//    csric0<float>("/lf_tool/matrix_cuda/coo5",4,4);
//    eigenvalue();
    eigenvalue2<float>();

	return 0;
}
